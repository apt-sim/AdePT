#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2020 CERN
// SPDX-License-Identifier: Apache-2.0

/// \file Raytracer.cu
/// \author Guilherme Amadio. Rewritten to use navigation from common code by Andrei Gheata.
/// Adapted from VecGeom for AdePT by antonio.petre@spacescience.ro

#include "Raytracer.h"
#include "RaytraceBenchmark.hpp"

#include <CopCore/Global.h>
#include <AdePT/BlockData.h>
#include <AdePT/LoopNavigator.h>

#include <VecGeom/base/Transformation3D.h>
#include <VecGeom/management/GeoManager.h>
#include <VecGeom/management/CudaManager.h>
#include <VecGeom/navigation/NavStateIndex.h>
#include <VecGeom/volumes/PlacedVolume.h>
#include <VecGeom/base/Stopwatch.h>
#include <VecGeom/base/Global.h>

#include <cassert>
#include <cstdio>

__global__ void RenderTile(adept::BlockData<Ray_t> *rays, RaytracerData_t rtdata, int offset_x, int offset_y,
                           int tile_size_x, int tile_size_y, unsigned char *tile_in, unsigned char *tile_out)
{
  int local_px = threadIdx.x + blockIdx.x * blockDim.x;
  int local_py = threadIdx.y + blockIdx.y * blockDim.y;

  if (local_px >= tile_size_x || local_py >= tile_size_y) return;

  int ray_index   = local_py * tile_size_x + local_px;
  int pixel_index = 4 * (local_py * tile_size_x + local_px);

  int global_px = offset_x + local_px;
  int global_py = offset_y + local_py;

  Ray_t *ray = (Ray_t *)(tile_in + ray_index * sizeof(Ray_t));
  ray->index = ray_index;

  (*rays)[ray_index]         = *ray;
  adept::Color_t pixel_color = Raytracer::RaytraceOne(rtdata, rays, global_px, global_py, ray->index);

  tile_out[pixel_index + 0] = pixel_color.fComp.red;
  tile_out[pixel_index + 1] = pixel_color.fComp.green;
  tile_out[pixel_index + 2] = pixel_color.fComp.blue;
  tile_out[pixel_index + 3] = 255;
}

// subdivide image in 16 tiles and launch each tile on a separate CUDA stream
void RenderTiledImage(adept::BlockData<Ray_t> *rays, cuda::RaytracerData_t *rtdata, NavIndex_t *output_buffer,
                      int block_size)
{
  hipStream_t streams[4];

  unsigned char *tile_host[16];
  unsigned char *tile_device_in[16];
  unsigned char *tile_device_out[16];

  int tile_size_x = rtdata->fSize_px / 4 + 1;
  int tile_size_y = rtdata->fSize_py / 4 + 1;

  // subdivide each tile in block_size x block_size thread blocks
  dim3 threads(block_size, block_size);
  dim3 blocks(tile_size_x / block_size + 1, tile_size_y / block_size + 1);

  for (int i = 0; i < 4; ++i)
    COPCORE_CUDA_CHECK(hipStreamCreate(&streams[i]));

  for (int i = 0; i < 16; ++i) {
    // allocate tile on host and device
    COPCORE_CUDA_CHECK(hipMalloc((void **)&tile_device_in[i], tile_size_x * tile_size_y * sizeof(cuda::Ray_t)));
    COPCORE_CUDA_CHECK(hipMalloc((void **)&tile_device_out[i], 4 * tile_size_x * tile_size_y));
    // CUDA streams require host memory to be pinned
    COPCORE_CUDA_CHECK(hipHostMalloc(&tile_host[i], 4 * tile_size_x * tile_size_y));
  }

  // wait for memory to reach GPU before launching kernels
  COPCORE_CUDA_CHECK(hipDeviceSynchronize());

  // call kernels to render each tile
  for (int ix = 0; ix < 4; ++ix) {
    for (int iy = 0; iy < 4; ++iy) {
      int idx      = 4 * ix + iy;
      int offset_x = ix * tile_size_x;
      int offset_y = iy * tile_size_y;

      RenderTile<<<blocks, threads, 0, streams[iy]>>>(rays, *rtdata, offset_x, offset_y, tile_size_x, tile_size_y,
                                                      tile_device_in[idx], tile_device_out[idx]);
    }
  }

  COPCORE_CUDA_CHECK(hipDeviceSynchronize());

  // copy back rendered tile to system memory
  for (int ix = 0; ix < 4; ++ix) {
    for (int iy = 0; iy < 4; ++iy) {
      int idx = 4 * ix + iy;
      COPCORE_CUDA_CHECK(hipMemcpyAsync(tile_host[idx], tile_device_out[idx], (size_t)4 * tile_size_x * tile_size_y,
                                         hipMemcpyDeviceToHost, streams[iy]));
      COPCORE_CUDA_CHECK(hipFree(tile_device_in[idx]));
      COPCORE_CUDA_CHECK(hipFree(tile_device_out[idx]));
    }
  }

  // ensure all tiles have been copied back
  COPCORE_CUDA_CHECK(hipDeviceSynchronize());
  COPCORE_CUDA_CHECK(hipGetLastError());

  for (int ix = 0; ix < 4; ++ix) {
    for (int iy = 0; iy < 4; ++iy) {
      int idx      = 4 * ix + iy;
      int offset_x = ix * tile_size_x;
      int offset_y = iy * tile_size_y;

      // copy each tile into the final destination
      for (int i = 0; i < tile_size_x; ++i) {
        for (int j = 0; j < tile_size_y; ++j) {
          int px          = offset_x + i;
          int py          = offset_y + j;
          int tile_index  = 4 * (j * tile_size_x + i);
          int pixel_index = 4 * (py * rtdata->fSize_px + px);

          if ((px >= rtdata->fSize_px) || (py >= rtdata->fSize_py)) continue;

          output_buffer[pixel_index + 0] = tile_host[idx][tile_index + 0];
          output_buffer[pixel_index + 1] = tile_host[idx][tile_index + 1];
          output_buffer[pixel_index + 2] = tile_host[idx][tile_index + 2];
          output_buffer[pixel_index + 3] = tile_host[idx][tile_index + 3];
        }
      }
      COPCORE_CUDA_CHECK(hipHostFree(tile_host[idx]));
    }
  }
  COPCORE_CUDA_CHECK(hipGetLastError());
}

void initiliazeCudaWorld(cuda::RaytracerData_t *rtdata) {
  
  // Load and synchronize the geometry on the GPU
  auto &cudaManager = vecgeom::cxx::CudaManager::Instance();
  cudaManager.LoadGeometry((vecgeom::cxx::VPlacedVolume *)rtdata->fWorld);
  cudaManager.Synchronize();

  auto gpu_world = cudaManager.world_gpu();
  assert(gpu_world && "GPU world volume is a null pointer");

  // Initialize the navigation state for the view point
  vecgeom::NavStateIndex vpstate;
  LoopNavigator::LocatePointIn(rtdata->fWorld, rtdata->fStart, vpstate, true);
  rtdata->fVPstate = vpstate;
  rtdata->fWorld   = gpu_world;

}

int executePipelineGPU(const vecgeom::cxx::VPlacedVolume *world, int argc, char *argv[])
{
  int result;
  result = runSimulation<copcore::BackendType::CUDA>(world, argc, argv);
  return result;
}

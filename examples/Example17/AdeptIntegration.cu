#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2022 CERN
// SPDX-License-Identifier: Apache-2.0

#include "AdeptIntegration.h"
#include "AdeptIntegration.cuh"

#include <VecGeom/base/Config.h>
#ifdef VECGEOM_ENABLE_CUDA
#include <VecGeom/backend/cuda/Interface.h>
#endif

#include <AdePT/Atomic.h>
#include <AdePT/BVHNavigator.h>
#include <AdePT/MParray.h>

#include <CopCore/Global.h>
#include <CopCore/PhysicalConstants.h>
#include <CopCore/Ranluxpp.h>

#include <G4Threading.hh>
#include <G4TransportationManager.hh>
#include <G4UniformMagField.hh>
#include <G4FieldManager.hh>

#include <G4HepEmState.hh>
#include <G4HepEmData.hh>
#include <G4HepEmState.hh>
#include <G4HepEmStateInit.hh>
#include <G4HepEmParameters.hh>
#include <G4HepEmMatCutData.hh>

#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <numeric>
#include <algorithm>

#include "electrons.cuh"
#include "gammas.cuh"

__constant__ __device__ struct G4HepEmParameters g4HepEmPars;
__constant__ __device__ struct G4HepEmData g4HepEmData;

__constant__ __device__ adeptint::VolAuxData *gVolAuxData = nullptr;
__constant__ __device__ double BzFieldValue               = 0;

G4HepEmState *AdeptIntegration::fg4hepem_state{nullptr};
int AdeptIntegration::kCapacity = 1024 * 1024;

void AdeptIntegration::VolAuxArray::InitializeOnGPU()
{
  // Transfer volume auxiliary data
  COPCORE_CUDA_CHECK(hipMalloc(&fAuxData_dev, sizeof(VolAuxData) * fNumVolumes));
  COPCORE_CUDA_CHECK(hipMemcpy(fAuxData_dev, fAuxData, sizeof(VolAuxData) * fNumVolumes, hipMemcpyHostToDevice));
  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gVolAuxData), &fAuxData_dev, sizeof(VolAuxData *)));
}

void AdeptIntegration::VolAuxArray::FreeGPU()
{
  COPCORE_CUDA_CHECK(hipFree(fAuxData_dev));
}

static G4HepEmState *InitG4HepEm()
{
  auto state = new G4HepEmState;
  InitG4HepEmState(state);

  G4HepEmMatCutData *cutData = state->fData->fTheMatCutData;
  G4cout << "fNumG4MatCuts = " << cutData->fNumG4MatCuts << ", fNumMatCutData = " << cutData->fNumMatCutData << G4endl;

  // Copy to GPU.
  CopyG4HepEmDataToGPU(state->fData);
  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g4HepEmPars), state->fParameters, sizeof(G4HepEmParameters)));

  // Create G4HepEmData with the device pointers.
  G4HepEmData dataOnDevice;
  dataOnDevice.fTheMatCutData   = state->fData->fTheMatCutData_gpu;
  dataOnDevice.fTheMaterialData = state->fData->fTheMaterialData_gpu;
  dataOnDevice.fTheElementData  = state->fData->fTheElementData_gpu;
  dataOnDevice.fTheElectronData = state->fData->fTheElectronData_gpu;
  dataOnDevice.fThePositronData = state->fData->fThePositronData_gpu;
  dataOnDevice.fTheSBTableData  = state->fData->fTheSBTableData_gpu;
  dataOnDevice.fTheGammaData    = state->fData->fTheGammaData_gpu;
  // The other pointers should never be used.
  dataOnDevice.fTheMatCutData_gpu   = nullptr;
  dataOnDevice.fTheMaterialData_gpu = nullptr;
  dataOnDevice.fTheElementData_gpu  = nullptr;
  dataOnDevice.fTheElectronData_gpu = nullptr;
  dataOnDevice.fThePositronData_gpu = nullptr;
  dataOnDevice.fTheSBTableData_gpu  = nullptr;
  dataOnDevice.fTheGammaData_gpu    = nullptr;

  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g4HepEmData), &dataOnDevice, sizeof(G4HepEmData)));

  return state;
}

// Kernel function to initialize tracks comming from a Geant4 buffer
__global__ void InitTracks(adeptint::TrackData *trackinfo, int ntracks, int startTrack, int event,
                           Secondaries secondaries, const vecgeom::VPlacedVolume *world, AdeptScoring *userScoring)
{
  constexpr double tolerance = 10. * vecgeom::kTolerance;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ntracks; i += blockDim.x * gridDim.x) {
    adept::TrackManager<Track> *trackmgr = nullptr;
    // These tracks come from Geant4, do not count them here
    switch (trackinfo[i].pdg) {
    case 11:
      trackmgr = secondaries.electrons;
      break;
    case -11:
      trackmgr = secondaries.positrons;
      break;
    case 22:
      trackmgr = secondaries.gammas;
    };
    assert(trackmgr != nullptr && "Unsupported pdg type");

    Track &track = trackmgr->NextTrack();
    track.rngState.SetSeed(1234567 * event + startTrack + i);
    track.energy       = trackinfo[i].energy;
    track.numIALeft[0] = -1.0;
    track.numIALeft[1] = -1.0;
    track.numIALeft[2] = -1.0;

    track.initialRange       = -1.0;
    track.dynamicRangeFactor = -1.0;
    track.tlimitMin          = -1.0;

    track.pos = {trackinfo[i].position[0], trackinfo[i].position[1], trackinfo[i].position[2]};
    track.dir = {trackinfo[i].direction[0], trackinfo[i].direction[1], trackinfo[i].direction[2]};
    track.navState.Clear();
    // We locate the pushed point because we run the risk that the
    // point is not located in the GPU region
    BVHNavigator::LocatePointIn(world, track.pos + tolerance * track.dir, track.navState, true);
    // The track must be on boundary at this point
    track.navState.SetBoundaryState(true);
    // nextState is initialized as needed.
    auto volume                         = track.navState.Top();
    int lvolID                          = volume->GetLogicalVolume()->id();
    adeptint::VolAuxData const &auxData = userScoring->GetAuxData_dev(lvolID);
    assert(auxData.fGPUregion);
  }
}

// Kernel to initialize the set of leaked queues per particle type.
__global__ void InitLeakedQueues(AllTrackManagers allMgr, size_t Capacity)
{
  for (int i = 0; i < ParticleType::NumParticleTypes; i++)
    MParrayTracks::MakeInstanceAt(Capacity, allMgr.leakedTracks[i]);
}

// Copy particles leaked from the GPU region into a compact buffer
__global__ void FillFromDeviceBuffer(int numLeaked, LeakedTracks all, adeptint::TrackData *fromDevice)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numLeaked) return;
  int numElectrons = all.leakedElectrons->size();
  int numPositrons = all.leakedPositrons->size();
  int numGammas    = all.leakedGammas->size();
  assert(numLeaked == numElectrons + numPositrons + numGammas);

  if (i < numElectrons) {
    fromDevice[i] = (*all.leakedElectrons)[i];
  } else if (i < numElectrons + numPositrons) {
    fromDevice[i] = (*all.leakedPositrons)[i - numElectrons];
  } else {
    fromDevice[i] = (*all.leakedGammas)[i - numElectrons - numPositrons];
  }
}

// Finish iteration: refresh track managers and fill statistics.
__global__ void FinishIteration(AllTrackManagers all, Stats *stats)
{
  for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
    all.trackmgr[i]->refresh_stats();
    stats->mgr_stats[i] = all.trackmgr[i]->fStats;
    stats->leakedTracks[i] = all.leakedTracks[i]->size();
  }
}

// Clear device leaked queues
__global__ void ClearLeakedQueues(LeakedTracks all)
{
  all.leakedElectrons->clear();
  all.leakedPositrons->clear();
  all.leakedGammas->clear();
}

bool AdeptIntegration::InitializeGeometry(const vecgeom::cxx::VPlacedVolume *world)
{
  // Upload geometry to GPU.
  auto &cudaManager = vecgeom::cxx::CudaManager::Instance();
  cudaManager.LoadGeometry(world);
  auto world_dev = cudaManager.Synchronize();
  // Initialize BVH
  InitBVH();

  return (world_dev != nullptr);
}

bool AdeptIntegration::InitializePhysics()
{
  // Initialize shared physics data
  AdeptIntegration::fg4hepem_state = InitG4HepEm();
  // Initialize field
  double bz = 0;
  auto field =
      (G4UniformMagField *)G4TransportationManager::GetTransportationManager()->GetFieldManager()->GetDetectorField();
  if (field) {
    auto field_vect = field->GetConstantFieldValue();
    bz              = field_vect[2];
  }
  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(BzFieldValue), &bz, sizeof(double)));

  return true;
}

void AdeptIntegration::PrepareLeakedBuffers(int numLeaked)
{
  // Make sure the size of the allocated track array is large enough
  using TrackData    = adeptint::TrackData;
  GPUstate &gpuState = *static_cast<GPUstate *>(fGPUstate);
  if (fBuffer.buffSize < numLeaked) {
    if (fBuffer.buffSize) {
      delete[] fBuffer.fromDeviceBuff;
      COPCORE_CUDA_CHECK(hipFree(gpuState.fromDevice_dev));
    }
    fBuffer.buffSize = numLeaked;
    fBuffer.fromDevice.reserve(numLeaked);
    fBuffer.fromDeviceBuff = new TrackData[numLeaked];
    COPCORE_CUDA_CHECK(hipMalloc(&gpuState.fromDevice_dev, numLeaked * sizeof(TrackData)));
  }
}

void AdeptIntegration::InitializeGPU()
{
  using TrackData    = adeptint::TrackData;
  fGPUstate          = new GPUstate;
  GPUstate &gpuState = *fGPUstate;

  // Allocate track managers, streams and synchronizaion events.
  const size_t QueueSize = MParrayTracks::SizeOfInstance(kCapacity);
  // Create a stream to synchronize kernels of all particle types.
  COPCORE_CUDA_CHECK(hipStreamCreate(&gpuState.stream));

  for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
    gpuState.allmgr_h.trackmgr[i]  = new adept::TrackManager<Track>(kCapacity);
    gpuState.allmgr_d.trackmgr[i]  = gpuState.allmgr_h.trackmgr[i]->ConstructOnDevice();
    gpuState.particles[i].trackmgr = gpuState.allmgr_d.trackmgr[i];
    COPCORE_CUDA_CHECK(hipMalloc(&gpuState.allmgr_d.leakedTracks[i], QueueSize));
    gpuState.particles[i].leakedTracks = gpuState.allmgr_d.leakedTracks[i];

    COPCORE_CUDA_CHECK(hipStreamCreate(&gpuState.particles[i].stream));
    COPCORE_CUDA_CHECK(hipEventCreate(&gpuState.particles[i].event));
  }
  InitLeakedQueues<<<1, 1, 0, gpuState.stream>>>(gpuState.allmgr_d, QueueSize);
  COPCORE_CUDA_CHECK(hipDeviceSynchronize());

  // initialize statistics
  COPCORE_CUDA_CHECK(hipMalloc(&gpuState.stats_dev, sizeof(Stats)));
  COPCORE_CUDA_CHECK(hipHostMalloc(&gpuState.stats, sizeof(Stats)));

  // initialize buffers of tracks on device
  COPCORE_CUDA_CHECK(hipMalloc(&gpuState.toDevice_dev, fMaxBatch * sizeof(TrackData)));
  PrepareLeakedBuffers(1000);
}

void AdeptIntegration::FreeGPU()
{
  // Free resources.
  GPUstate &gpuState = *static_cast<GPUstate *>(fGPUstate);
  COPCORE_CUDA_CHECK(hipFree(gpuState.stats_dev));
  COPCORE_CUDA_CHECK(hipHostFree(gpuState.stats));
  COPCORE_CUDA_CHECK(hipFree(gpuState.toDevice_dev));

  COPCORE_CUDA_CHECK(hipStreamDestroy(gpuState.stream));

  for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
    gpuState.allmgr_h.trackmgr[i]->FreeFromDevice();
    delete gpuState.allmgr_h.trackmgr[i];
    COPCORE_CUDA_CHECK(hipFree(gpuState.particles[i].leakedTracks));

    COPCORE_CUDA_CHECK(hipStreamDestroy(gpuState.particles[i].stream));
    COPCORE_CUDA_CHECK(hipEventDestroy(gpuState.particles[i].event));
  }

  // Free G4HepEm data
  FreeG4HepEmData(AdeptIntegration::fg4hepem_state->fData);
  delete AdeptIntegration::fg4hepem_state;
  AdeptIntegration::fg4hepem_state = nullptr;
}

void AdeptIntegration::ShowerGPU(int event, TrackBuffer &buffer) // const &buffer)
{
  using TrackData = adeptint::TrackData;
  // Capacity of the different containers aka the maximum number of particles.
  auto &cudaManager                             = vecgeom::cxx::CudaManager::Instance();
  COPCORE_CUDA_CHECK(vecgeom::cxx::CudaDeviceSetStackLimit(8192));
  const vecgeom::cuda::VPlacedVolume *world_dev = cudaManager.world_gpu();
  GPUstate &gpuState                            = *static_cast<GPUstate *>(fGPUstate);

  Secondaries secondaries{gpuState.allmgr_d.trackmgr[0], gpuState.allmgr_d.trackmgr[1], gpuState.allmgr_d.trackmgr[2]};

  ParticleType &electrons = gpuState.particles[ParticleType::Electron];
  ParticleType &positrons = gpuState.particles[ParticleType::Positron];
  ParticleType &gammas    = gpuState.particles[ParticleType::Gamma];

  // copy buffer of tracks to device
  COPCORE_CUDA_CHECK(hipMemcpyAsync(gpuState.toDevice_dev, buffer.toDevice.data(),
                                     buffer.toDevice.size() * sizeof(adeptint::TrackData), hipMemcpyHostToDevice,
                                     gpuState.stream));

  if (fDebugLevel > 0) {
    G4cout << std::dec << G4endl << "GPU transporting event " << event << " for CPU thread "
           << G4Threading::G4GetThreadId() << ": " << std::flush;
  }

  // Initialize AdePT tracks using the track buffer copied from CPU
  constexpr int initThreads = 32;
  int initBlocks            = (buffer.toDevice.size() + initThreads - 1) / initThreads;

  InitTracks<<<initBlocks, initThreads, 0, gpuState.stream>>>(
      gpuState.toDevice_dev, buffer.toDevice.size(), buffer.startTrack, event, secondaries, world_dev, fScoring_dev);

  COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));

  gpuState.allmgr_h.trackmgr[ParticleType::Electron]->fStats.fInFlight = buffer.nelectrons;
  gpuState.allmgr_h.trackmgr[ParticleType::Positron]->fStats.fInFlight = buffer.npositrons;
  gpuState.allmgr_h.trackmgr[ParticleType::Gamma]->fStats.fInFlight    = buffer.ngammas;

  constexpr float compactThreshold = 0.9;
  constexpr int MaxBlocks          = 1024;
  constexpr int TransportThreads   = 32;
  int transportBlocks;

  int inFlight          = 0;
  int killed            = 0;
  int numLeaked         = 0;
  int num_compact       = 0;
  int loopingNo         = 0;
  int previousElectrons = -1, previousPositrons = -1;
  LeakedTracks leakedTracks = {.leakedElectrons = electrons.leakedTracks,
                            .leakedPositrons = positrons.leakedTracks,
                            .leakedGammas    = gammas.leakedTracks};

  auto copyLeakedTracksFromGPU = [&](int numLeaked)
  {
    PrepareLeakedBuffers(numLeaked);
    // Populate the buffer from sparse memory
    constexpr unsigned int block_size = 256;
    unsigned int grid_size            = (numLeaked + block_size - 1) / block_size;
    FillFromDeviceBuffer<<<grid_size, block_size, 0, gpuState.stream>>>(numLeaked, leakedTracks,
                                                                        gpuState.fromDevice_dev);
    // Copy the buffer from device to host
    COPCORE_CUDA_CHECK(hipMemcpyAsync(fBuffer.fromDeviceBuff, gpuState.fromDevice_dev, numLeaked * sizeof(TrackData),
                                        hipMemcpyDeviceToHost, gpuState.stream));
    COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));
    fBuffer.fromDevice.insert(fBuffer.fromDevice.end(), &fBuffer.fromDeviceBuff[0], &fBuffer.fromDeviceBuff[numLeaked]);
  };
  
  int niter = 0;
  do {

    // *** ELECTRONS ***
    int numElectrons = gpuState.allmgr_h.trackmgr[ParticleType::Electron]->fStats.fInFlight;
    if (numElectrons > 0) {
      transportBlocks = (numElectrons + TransportThreads - 1) / TransportThreads;
      transportBlocks = std::min(transportBlocks, MaxBlocks);

      TransportElectrons<AdeptScoring><<<transportBlocks, TransportThreads, 0, electrons.stream>>>(
          electrons.trackmgr, secondaries, electrons.leakedTracks, fScoring_dev);

      COPCORE_CUDA_CHECK(hipEventRecord(electrons.event, electrons.stream));
      COPCORE_CUDA_CHECK(hipStreamWaitEvent(gpuState.stream, electrons.event, 0));
    }

    // *** POSITRONS ***
    int numPositrons = gpuState.allmgr_h.trackmgr[ParticleType::Positron]->fStats.fInFlight;
    if (numPositrons > 0) {
      transportBlocks = (numPositrons + TransportThreads - 1) / TransportThreads;
      transportBlocks = std::min(transportBlocks, MaxBlocks);

      TransportPositrons<AdeptScoring><<<transportBlocks, TransportThreads, 0, positrons.stream>>>(
          positrons.trackmgr, secondaries, positrons.leakedTracks, fScoring_dev);

      COPCORE_CUDA_CHECK(hipEventRecord(positrons.event, positrons.stream));
      COPCORE_CUDA_CHECK(hipStreamWaitEvent(gpuState.stream, positrons.event, 0));
    }

    // *** GAMMAS ***
    int numGammas = gpuState.allmgr_h.trackmgr[ParticleType::Gamma]->fStats.fInFlight;
    if (numGammas > 0) {
      transportBlocks = (numGammas + TransportThreads - 1) / TransportThreads;
      transportBlocks = std::min(transportBlocks, MaxBlocks);

      TransportGammas<AdeptScoring><<<transportBlocks, TransportThreads, 0, gammas.stream>>>(
          gammas.trackmgr, secondaries, gammas.leakedTracks, fScoring_dev);

      COPCORE_CUDA_CHECK(hipEventRecord(gammas.event, gammas.stream));
      COPCORE_CUDA_CHECK(hipStreamWaitEvent(gpuState.stream, gammas.event, 0));
    }

    // *** END OF TRANSPORT ***

    // The events ensure synchronization before finishing this iteration and
    // copying the Stats back to the host.
    FinishIteration<<<1, 1, 0, gpuState.stream>>>(gpuState.allmgr_d, gpuState.stats_dev);
    COPCORE_CUDA_CHECK(
        hipMemcpyAsync(gpuState.stats, gpuState.stats_dev, sizeof(Stats), hipMemcpyDeviceToHost, gpuState.stream));

    // Finally synchronize all kernels.
    COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));

    // Count the number of particles in flight.
    inFlight  = 0;
    numLeaked = 0;
    for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
      // Update stats for host track manager objects
      gpuState.allmgr_h.trackmgr[i]->fStats = gpuState.stats->mgr_stats[i];
      inFlight += gpuState.stats->mgr_stats[i].fInFlight;
      numLeaked += gpuState.stats->leakedTracks[i];
      // Compact the particle track buffer if needed
      auto compacted = gpuState.allmgr_h.trackmgr[i]->SwapAndCompact(compactThreshold, gpuState.particles[i].stream);
      if (compacted) num_compact++;
    }

    // Check if only charged particles are left that are looping.
    numElectrons = gpuState.allmgr_h.trackmgr[ParticleType::Electron]->fStats.fInFlight;
    numPositrons = gpuState.allmgr_h.trackmgr[ParticleType::Positron]->fStats.fInFlight;
    numGammas    = gpuState.allmgr_h.trackmgr[ParticleType::Gamma]->fStats.fInFlight;
    if (fDebugLevel > 1) {
      printf("iter %d: elec %d, pos %d, gam %d, leak %d\n", niter++, numElectrons, numPositrons, numGammas, numLeaked);
    }
    if (numElectrons == previousElectrons && numPositrons == previousPositrons && numGammas == 0) {
      loopingNo++;
    } else {
      previousElectrons = numElectrons;
      previousPositrons = numPositrons;
      loopingNo         = 0;
    }

  } while (inFlight > 0 && loopingNo < 200);

  if (fDebugLevel > 0) {
    G4cout << inFlight << " in flight, " << numLeaked << " leaked, " << num_compact << " compacted\n";
  }

  // Transfer the leaked tracks from GPU
  if (numLeaked) {
    copyLeakedTracksFromGPU(numLeaked);
    // Sort by energy the tracks coming from device to ensure reproducibility
    std::sort(fBuffer.fromDevice.begin(), fBuffer.fromDevice.end());
  }

  if (inFlight > 0) {
    killed += inFlight;
    for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
      int inFlightParticles = gpuState.allmgr_h.trackmgr[i]->fStats.fInFlight;
      if (inFlightParticles == 0) {
        continue;
      }

      gpuState.allmgr_h.trackmgr[i]->Clear(gpuState.particles[i].stream);
    }
    COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));
  }

  ClearLeakedQueues<<<1, 1, 0, gpuState.stream>>>(leakedTracks);
  COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));

  // Transfer back scoring.
  fScoring->CopyHitsToHost();
  fScoring->fGlobalScoring.numKilled = inFlight;
}

#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2020 CERN
// SPDX-License-Identifier: Apache-2.0

// Author: J. Apostolakis  12 Nov 2020

#include <cstdio>
#include <iomanip>

#include <CopCore/SystemOfUnits.h>
#include <CopCore/PhysicalConstants.h>

// #include <CopCore/Ranluxpp.h>

#include <VecGeom/base/Vector3D.h>
#include "track.h"

#include <AdePT/BlockData.h>

#include "uniformMagField.h"
#include "fieldPropagator.h"

#include "trackBlock.h"
// using trackBlock_t  = adept::BlockData<track>;

using copcore::units::meter;
using copcore::units::GeV;
using copcore::units::MeV;

constexpr double  minX = -2.0 * meter, maxX = 2.0 * meter;
constexpr double  minY = -3.0 * meter, maxY = 3.0 * meter;
constexpr double  minZ = -5.0 * meter, maxZ = 5.0 * meter;

// constexpr double  maxP = 1.0 * GeV;

constexpr double maxStepSize = 0.1 * ( (maxX - minX) + (maxY - minY) + (maxZ - minZ) );

#include <CopCore/Ranluxpp.h>

__device__ void initOneTrack(unsigned int  index,
                             uint64_t      rngBase,
                             track       & aTrack,
                             unsigned int  eventId
   )
{
  // Very basic initial state for RNG ... to be improved
  aTrack.rng_state.SetSeed( rngBase + (uint64_t) index);
   
  float r = aTrack.uniform(); // hiprand_uniform(states);  
  // aTrack.charge = ( r < 0.45 ? -1 : ( r< 0.9 ? 0 : +1 ) );
  constexpr  int  pdgElec = 11 , pdgGamma = 22;
  aTrack.pdg = ( r < 0.45 ? pdgElec : ( r< 0.9 ? pdgGamma : -pdgElec ) );

  // Make the first tracks electrons -- for now
  if( index < 20 ) aTrack.pdg = pdgElec;
  
  aTrack.index = index;
  aTrack.eventId = eventId;
  
  aTrack.pos[0] = 0.0; // minX + aTrack.uniform() * ( maxX - minX );
  aTrack.pos[1] = 0.0; // minY + aTrack.uniform() * ( maxY - minY );
  aTrack.pos[2] = 0.0; // minZ + aTrack.uniform() * ( maxZ - minZ );

  double  px, py, pz;
  px = 4 * MeV ; // maxP * 2.0 * ( aTrack.uniform() - 0.5 );   // -maxP to +maxP
  py = 0; // maxP * 2.0 * ( aTrack.uniform() - 0.5 );
  pz = 3 * MeV ; // maxP * 2.0 * ( aTrack.uniform() - 0.5 );

  double  pmag2 =  px*px + py*py + pz*pz;
  double  inv_pmag = 1.0 / std::sqrt(pmag2);
  aTrack.dir[0] = px * inv_pmag; 
  aTrack.dir[1] = py * inv_pmag; 
  aTrack.dir[2] = pz * inv_pmag;

  aTrack.interaction_length = 0.001 * (index+1) * maxStepSize ; // aTrack.uniform() * maxStepSize;
  
  // double  mass = ( aTrack.pdg == pdgGamma ) ?  0.0 : kElectronMassC2 ; // rest mass
  double  mass = aTrack.mass();
  aTrack.energy = pmag2 / ( sqrt( mass * mass + pmag2 ) + mass);
  // More accurate than   ( sqrt( mass * mass + pmag2 ) - mass);
}

// this GPU kernel function is used to create and initialize 
//     .. the particles' state 

__global__ void initTracks( adept::BlockData<track> *trackBlock,
                            unsigned int numTracks,
                            unsigned int eventId,
                            unsigned int   runId = 0
                          )
{
  /* initialize the tracks with random particles */
  int pclIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (pclIdx >= numTracks) return;

  track* pTrack =   trackBlock->NextElement();

  uint64_t  rngBase =     runId * (uint64_t(1)<<52)
                      + eventId * (uint64_t(1)<<36);

  initOneTrack( pclIdx, rngBase, *pTrack, eventId );
}

__global__ void overwriteTracks( adept::BlockData<track> *trackBlock,
                                 unsigned int numTracks,
                                 unsigned int eventId,
                                 unsigned int   runId = 0                                 
   )
{
  /* initialize the tracks with random particles */
  int pclIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if( trackBlock->GetNholes() > 0 ) return;
  // We can only overwrite if there are no holes !
  
  numTracks = max ( numTracks, (unsigned int) trackBlock->GetNused() );
  if (pclIdx >= numTracks ) return;
  
  track & trk = (*trackBlock)[pclIdx];
  uint64_t  rngBase =     runId * (uint64_t(1)<<52)
                      + eventId * (uint64_t(1)<<36);
  
  initOneTrack( pclIdx, rngBase, (*trackBlock)[pclIdx], eventId );
}

     
static float BzValue = 0.1 * copcore::units::tesla;

static float BfieldValue[3] = { 0.001 * copcore::units::tesla,
                               -0.001 * copcore::units::tesla,
                               BzValue };

// V1 -- field along Z axis
__global__ void fieldPropagatorBz_glob(adept::BlockData<track> *trackBlock, float Bz )
{
  vecgeom::Vector3D<double> endPosition;
  vecgeom::Vector3D<double> endDirection;

  int maxIndex = trackBlock->GetNused() + trackBlock->GetNholes();   
  
  // Non-block version:
  //   int pclIdx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pclIdx  = blockIdx.x * blockDim.x + threadIdx.x;  pclIdx < maxIndex;
           pclIdx += blockDim.x * gridDim.x)
  {
     track& aTrack= (*trackBlock)[pclIdx];

     // check if you are not outside the used block
     if (pclIdx >= maxIndex || aTrack.status == dead) continue;

     fieldPropagatorConstBz(aTrack, Bz, endPosition, endDirection);

     // Update position, direction     
     aTrack.pos = endPosition;  
     aTrack.dir = endDirection;
  }
}

// V2 -- constant field any direction 
__global__ void fieldPropagatorAnyDir_glob(adept::BlockData<track> *trackBlock,
                                           // float Bx, float By, float Bz,
                                           uniformMagField Bfield )  // by value !?
                                           // const uniformMagField& Bfield )   
{
  // template <type T> using Vector3D = vecgeom::Vector3D<T>;
  vecgeom::Vector3D<double> endPosition;
  vecgeom::Vector3D<double> endDirection;
  
  int maxIndex = trackBlock->GetNused() + trackBlock->GetNholes();   

  float Bvalue[3];
  Bfield.ObtainField( Bvalue );
  
  ConstFieldHelixStepper helixAnyB=  ConstFieldHelixStepper( Bvalue );
  
  // Non-block version:
  //   int pclIdx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pclIdx  = blockIdx.x * blockDim.x + threadIdx.x;  pclIdx < maxIndex;
           pclIdx += blockDim.x * gridDim.x)
  {
     track& aTrack= (*trackBlock)[pclIdx];

     // check if you are not outside the used block
     if (pclIdx >= maxIndex || aTrack.status == dead) continue;

     fieldPropagatorConstBgeneral(aTrack, helixAnyB, endPosition, endDirection);

     // Update position, direction     
     aTrack.pos = endPosition;  
     aTrack.dir = endDirection;
  }
}

int main( int argc, char** argv )
{
  // template<type T>
  using ThreeVector = vecgeom::Vector3D<double>; 
  constexpr int numBlocks=2, numThreadsPerBlock=16;
  int  totalNumThreads = numBlocks * numThreadsPerBlock;
  bool useBzOnly = true;

  if( argc > 1 )
     useBzOnly = false;
  
  const int numTracks = totalNumThreads; // Constant at first ...

  std::cout << "Magnetic field used: " << std::endl;
  if( !useBzOnly ){
     std::cout << "  Bx = " << BfieldValue[0] / copcore::units::tesla << " T " << std::endl;
     std::cout << "  By = " << BfieldValue[1] / copcore::units::tesla << " T " << std::endl;
  } 
  std::cout << "  Bz = " << BzValue / copcore::units::tesla << " T " << std::endl;

  // uniformMagField Bfield( BfieldValue );
  uniformMagField BfieldObj( BfieldValue );
  
  // Track capacity of the block
  constexpr int capacity = 1 << 16;

  // 1. Create container of Tracks  BlockData<track>
  // -----------------------------------------------------
  std::cout << " Creating track buffer for " << capacity << " tracks -" // " on GPU device."
            << " in Unified Memory." 
            << std::endl;
  
  // Allocate a block of tracks with capacity larger than the total number of spawned threads
  // Note that if we want to allocate several consecutive block in a buffer, we have to use
  // Block_t::SizeOfAlignAware rather than SizeOfInstance to get the space needed per block
  size_t blocksize = trackBlock_t::SizeOfInstance(capacity);
  char *buffer2    = nullptr;
  hipError_t allocErr= hipMallocManaged(&buffer2, blocksize);  // Allocated in Unified memory ... (baby steps)

  // auto trackBlock_dev  = trackBlock_t::MakeInstanceAt(capacity, buffer2);  
  auto trackBlock_uniq = trackBlock_t::MakeInstanceAt(capacity, buffer2); // Unified memory => _uniq

  // 2.  Initialise track - on device
  // --------------------------------
  std::cout << " Initialising tracks." << std::endl;
  std::cout << " Max step size = " << maxStepSize << std::endl;

  unsigned  int runId= 101, eventId = 1;
  unsigned  int numTracksEv1 = numTracks / 2;
  initTracks<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, numTracksEv1, eventId, runId );
  initTracks<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, numTracks-numTracksEv1, ++eventId, runId );  
                                                
  hipDeviceSynchronize();

  const unsigned int SmallNum= std::max( 2, numTracks);

  std::cout << std::endl;
  std::cout << " Initialised tracks: " << std::endl;
  printTracks( trackBlock_uniq, false, numTracks );  

  overwriteTracks<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, numTracks, ++eventId, runId );
  hipDeviceSynchronize();  
  std::cout << " Overwritten tracks: " << std::endl;
  printTracks( trackBlock_uniq, false, numTracks );  
  
  // Copy to array for host to cross-check
  track tracksStart_host[SmallNum];  
  // memcpy(tracksStart_host, &(*trackBlock_uniq)[0], SmallNum*sizeof(track));
  for ( int i = 0; i < SmallNum ; i++ ){
     tracksStart_host[i] = (*trackBlock_uniq)[i];
     // Print copy to check 
     // std::cout << " Orig: ";  (*trackBlock_uniq)[i].print( i );
     // std::cout << " Copy: ";  (tracksStart_host[i]).print( i );
  }
  
  // Else if stored on device: 
  //  hipMemcpy(tracksStart_host, &(*trackBlock_dev)[0], SmallNum*sizeof(track), hipMemcpyDeviceToHost );
  
  // 3. Propagate tracks -- on device
  if( useBzOnly ){
     fieldPropagatorBz_glob<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, BzValue );
     //*********
  } else {
     fieldPropagatorAnyDir_glob<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq,
                                                                   BfieldObj );
  }
  hipDeviceSynchronize();  

  // 4. Check results on host
  std::cout << " Calling move in field (host)." << std::endl;

  vecgeom::Vector3D<float> magFieldVec( BfieldValue[0],
                                        BfieldValue[1],
                                        BfieldValue[2] );
  ConstFieldHelixStepper  helixStepper( magFieldVec); // -> BfieldObj );  // Re-use it (expensive sqrt & div.)
  
  for( int i = 0; i<SmallNum ; i++){
     ThreeVector endPosition, endDirection;
     track  hostTrack = tracksStart_host[i];  // (*trackBlock_uniq)[i];
     // hostTrack.pos = 

     if( useBzOnly ){     
        fieldPropagatorConstBz( hostTrack, BzValue, endPosition, endDirection );
     } else {
        fieldPropagatorConstBgeneral( hostTrack, helixStepper, endPosition, endDirection );        
     }
     
     double move       = (endPosition  - hostTrack.pos).Mag();
     double deflection = (endDirection - hostTrack.dir).Mag();
     
     // Update position, direction     
     hostTrack.pos = endPosition;  
     hostTrack.dir = endDirection;

     track  devTrackVal= (*trackBlock_uniq)[i];
     ThreeVector posDiff = hostTrack.pos - devTrackVal.pos;     
     ThreeVector dirDiff = hostTrack.dir - devTrackVal.dir;

     constexpr double tol = 1.0e-07;
     bool badPosition  = posDiff.Mag() > tol * move;
     bool badDirection = dirDiff.Mag() > tol * deflection;
     
     if( badPosition || badDirection ){
        std::cout << std::endl;        
        std::cout << " Difference seen for Track " << i
                  << " addr = " << & (*trackBlock_uniq)[i]
                  << std::endl;
        std::cout << std::endl;
        // std::cout << " Track " << i << " addr = " << &aTrack << std::endl;
        // std::cout << " Track " << i << " pdg = " << aTrack.pdg
        //          << " x,y,z = " << aTrack.position[0] << " , " << aTrack.position[1]
        //          << " , " << aTrack.position[3] << std::endl;
        std::cout << " Ref (host) = ";
        hostTrack.print( i );

        std::cout << " Device     = ";
        devTrackVal.print( i );

        if( badPosition ){
           std::cout << " Position  diff = " << posDiff << " mag = " << posDiff.Mag() << " vs move      = " << move << " " << std::endl;
        }
        if( badDirection ){
           std::cout << " Direction diff = " << dirDiff << " mag = " << dirDiff.Mag() << " vs deflection = " << deflection << " " << std::endl;
        }
        std::cout << std::endl;
        // printTrack( hostTrack, i );
     }
  }
  // std::cout << " Tracks moved in host: " << std::endl;
  // printTrackBlock( trackBlock_uniq, numTracks );

  std::cout << std::endl;
  std::cout << " Calling move in field (device)" << std::endl;

  int maxIndex = trackBlock_uniq->GetNused() + trackBlock_uniq->GetNholes();     
  std::cout  << " maxIndex = " << maxIndex
             << " numTracks = " << numTracks << std::endl;

  // 5.  Report result of movement
  // 
  //          See where they went ?
  std::cout << " Ending tracks: " << std::endl;
  printTracks( trackBlock_uniq, false, numTracks );
}


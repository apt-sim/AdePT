#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2020 CERN
// SPDX-License-Identifier: Apache-2.0

// Author: J. Apostolakis  12 Nov 2020

#include <cstdio>
#include <iomanip>

#include <CopCore/SystemOfUnits.h>
#include <CopCore/PhysicalConstants.h>

// #include <CopCore/Ranluxpp.h>

#include <VecGeom/base/Vector3D.h>
#include "track.h"

#include <AdePT/BlockData.h>

#include "ConstBzFieldStepper.h"

using floatX_t = double;  //  float type for X = position
using floatE_t = double;  //  float type for E = energy  & momentum

using  TrackBlock_t    = adept::BlockData<track>;

template<unsigned int N>
struct FieldPropagationBuffer
{
  int      charge[N];
  floatX_t position[3][N];
  floatE_t momentum[3][N];
  int      index[N];
  bool     active[N];
};

using copcore::units::kElectronMassC2;

using copcore::units::meter;
using copcore::units::GeV;
using copcore::units::MeV;

constexpr floatX_t  minX = -2.0 * meter, maxX = 2.0 * meter;
constexpr floatX_t  minY = -3.0 * meter, maxY = 3.0 * meter;
constexpr floatX_t  minZ = -5.0 * meter, maxZ = 5.0 * meter;

// constexpr floatE_t  maxP = 1.0 * GeV;

constexpr floatX_t maxStepSize = 0.1 * ( (maxX - minX) + (maxY - minY) + (maxZ - minZ) );

#include <CopCore/Ranluxpp.h>

__device__ void initOneTrack(unsigned int  index,
                             uint64_t      rngBase,
                             track       & aTrack
   )
{
  // Very basic initial state for RNG ... to be improved
  aTrack.rng_state.SetSeed( rngBase + (uint64_t) index);
   
  float r = aTrack.uniform(); // hiprand_uniform(states);  
  // aTrack.charge = ( r < 0.45 ? -1 : ( r< 0.9 ? 0 : +1 ) );
  constexpr  int  pdgElec = 11 , pdgGamma = 22;
  aTrack.pdg = ( r < 0.45 ? pdgElec : ( r< 0.9 ? pdgGamma : -pdgElec ) );

  aTrack.pos[0] = 0.0; // minX + aTrack.uniform() * ( maxX - minX );
  aTrack.pos[1] = 0.0; // minY + aTrack.uniform() * ( maxY - minY );
  aTrack.pos[2] = 0.0; // minZ + aTrack.uniform() * ( maxZ - minZ );

  floatE_t  px, py, pz;
  px = 4 * MeV ; // maxP * 2.0 * ( aTrack.uniform() - 0.5 );   // -maxP to +maxP
  py = 0; // maxP * 2.0 * ( aTrack.uniform() - 0.5 );
  pz = 3 * MeV ; // maxP * 2.0 * ( aTrack.uniform() - 0.5 );

  floatE_t  pmag2 =  px*px + py*py + pz*pz;
  floatE_t  inv_pmag = 1.0 / std::sqrt(pmag2);
  aTrack.dir[0] = px * inv_pmag; 
  aTrack.dir[1] = py * inv_pmag; 
  aTrack.dir[2] = pz * inv_pmag;

  aTrack.interaction_length = 0.001 * index * maxStepSize ; // aTrack.uniform() * maxStepSize;
  
  floatE_t  mass = ( aTrack.pdg == pdgGamma ) ?  0.0 : kElectronMassC2 ; // rest mass
  aTrack.energy = pmag2 / ( sqrt( mass * mass + pmag2 ) + mass);
}

// this GPU kernel function is used to initialize 
//     .. the particles' state ?

__global__ void initTracks( adept::BlockData<track> *trackBlock,
                            unsigned int numTracks,                            
                            unsigned int eventId,
                            unsigned int   runId = 101
                          )
{
  /* initialize the tracks with random particles */
  int pclIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (pclIdx >= numTracks) return;

  track* pTrack =   trackBlock->NextElement();

  uint64_t  rngBase =     runId * (uint64_t(1)<<52)
                      + eventId * (uint64_t(1)<<36);

  initOneTrack( pclIdx, rngBase, *pTrack ); // , &states[pclIdx] );
}

constexpr float BzValue = 0.1 * copcore::units::tesla; 

// VECCORE_ATT_HOST_DEVICE
__host__  __device__ 
void EvaluateField( const floatX_t pos[3], float fieldValue[3] )
{
    fieldValue[0]= 0.0;
    fieldValue[1]= 0.0;
    fieldValue[2]= BzValue;        
}

#ifdef USE_VECTOR3D
#include <VecGeom/Vector3D.h>
#endif

__host__ __device__
void moveInField(track& track)
{
  floatX_t  step= track.interaction_length;

  // Charge for e+ / e-  only    ( gamma / other neutrals also ok.) 
  int    charge = track.charge(); // (track.pdg == -11) - (track.pdg == 11);
  
  if ( charge == 0.0 ) return;
  
  // floatX_t pclPosition[3];

  // Evaluate initial field value
  // EvaluateField( pclPosition3d, fieldVector );

  // float restMass = ElectronMass;  // For now ... 
  floatE_t kinE = track.energy;
  floatE_t momentumMag = sqrt( kinE * ( kinE + 2.0 * kElectronMassC2) );
  
  // Collect position, momentum
  // floatE_t momentum[3] = { momentumMag * track.dir[0], 
  //                          momentumMag * track.dir[1], 
  //                          momentumMag * track.dir[2] } ;
#ifdef VECTOR3D    
  vecGeom::Vector3D<floatX_t> positionOut3d(  track.pos );
  vecGeom::Vector3D<floatX_t> directionOut3d( track.dir );
#endif
  
  ConstBzFieldStepper  helixBz(BzValue);

#if 0    
  track.pos[0] += 0.1 * ( 1. + 0.0001 * step );
  track.pos[1] += 0.2;
  track.pos[2] += 0.3;
  track.direction[0] += 0.3;
  track.direction[1] += 0.2;
  track.direction[2] += 0.1;
#endif    

  // For now all particles ( e-, e+, gamma ) can be propagated using this
  //   for gammas  charge = 0 works, and ensures that it goes straight.
#ifndef USE_VECTOR3D
  floatX_t xOut, yOut, zOut, dirX, dirY, dirZ;  
  helixBz.DoStep( track.pos[0], track.pos[1], track.pos[2],
                  track.dir[0], track.dir[1], track.dir[2],
                  charge, momentumMag, step,
                  xOut, yOut, zOut, dirX, dirY, dirZ );                  

  // Update position, direction
  track.pos[0] = xOut;
  track.pos[1] = yOut;
  track.pos[2] = zOut;
  track.dir[0] = dirX;
  track.dir[1] = dirY;
  track.dir[2] = dirZ;  
#else  
  helixBz.DoStep( track.pos, track.dir, charge, momentumMag, step,
                  positionOut3d, directionOut3d);

  // Update position, direction
  track.pos = positionOut3d;  
  // track.pos[0] = positionOut3d[0];
  // track.pos[1] = positionOut3d[1];
  // track.pos[2] = positionOut3d[2];
  track.dir = directionOut3d;
  // track.dir[0] = directionOut3d[0];
  // track.dir[1] = directionOut3d[1];
  // track.dir[2] = directionOut3d[2];
#endif

  // Alternative: load into local variables ?
  // float xIn= track.position[0], yIn= track.position[1], zIn = track.position[2];
  // float dirXin= track.direction[0], dirYin = track.direction[1], dirZin = track.direction[2];


}

// V1 -- one per warp
__global__ void moveInField_glob(adept::BlockData<track> *trackBlock, int numTracksChk )
{
  int maxIndex = trackBlock->GetNused() + trackBlock->GetNholes();   

  // Non-block version:
  //   int pclIdx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pclIdx  = blockIdx.x * blockDim.x + threadIdx.x;  pclIdx < maxIndex;
           pclIdx += blockDim.x * gridDim.x)
  {
     track &aTrack= (*trackBlock)[pclIdx];

     // check if you are not outside the used block
     if (pclIdx >= maxIndex || aTrack.status == dead) continue;
  
     moveInField(aTrack);
  }
}

void reportOneTrack( const track & aTrack, int id = -1 )
{
   using std::setw;
   
   std::cout << " Track " << setw(4) << id
             << " addr= " << & aTrack   << " "
             << " pdg = " << setw(4) << aTrack.pdg
             << " x,y,z = "
             << setw(12) << aTrack.pos[0] << " , "
             << setw(12) << aTrack.pos[1] << " , "
             << setw(12) << aTrack.pos[2]
             << " step = " << setw( 12 ) << aTrack.interaction_length
             << " kinE = " << setw( 10 ) << aTrack.energy
             << " Dir-x,y,z = "
             << setw(12) << aTrack.dir[0] << " , "
             << setw(12) << aTrack.dir[1] << " , "
             << setw(12) << aTrack.dir[2]
             << std::endl;
}

void reportTracks( TrackBlock_t* trackBlock, unsigned int numTracks )
{
  // unsigned int sizeOfTrack = TrackBlock_t::SizeOfAlignAware;
  // size_t  bytesForTracks   = TrackBlock_t::SizeOfInstance(numTracks);
  // mallocManaged(&buffer2, blocksize);
  
  // track tracksEnd_host[SmallNum];
  // hipMemcpy(tracksEnd_host, trackBlock_dev, SmallNum * sizeOfTrack, // sizeof(track),
  //            hipMemcpyDeviceToHost );

  // std::cout << " TrackBlock addr= " << trackBlock   << " " << std::endl;
  for( int i = 0; i<numTracks ; i++) {
     track& aTrack = (*trackBlock)[i];
     reportOneTrack( aTrack, i );
  }
}

int main( int argc, char** argv )
{
  constexpr int numBlocks=2, numThreadsPerBlock=16;
  int  totalNumThreads = numBlocks * numThreadsPerBlock;
  
  const int numTracks = totalNumThreads; // Constant at first ...
  
  std::cout << " Bz = " << BzValue / copcore::units::tesla << " T " << std::endl;
  
  // Track capacity of the block
  constexpr int capacity = 1 << 16;

  // 1. Create container of Tracks  BlockData<track>
  // -----------------------------------------------------
  std::cout << " Creating track buffer for " << capacity << " tracks -" // " on GPU device."
            << " in Unified Memory." 
            << std::endl;
  
  // Allocate a block of tracks with capacity larger than the total number of spawned threads
  // Note that if we want to allocate several consecutive block in a buffer, we have to use
  // Block_t::SizeOfAlignAware rather than SizeOfInstance to get the space needed per block
  size_t blocksize = TrackBlock_t::SizeOfInstance(capacity);
  char *buffer2    = nullptr;
  hipError_t allocErr= hipMallocManaged(&buffer2, blocksize);  // Allocated in Unified memory ... (baby steps)

  // auto trackBlock_dev  = TrackBlock_t::MakeInstanceAt(capacity, buffer2);  
  auto trackBlock_uniq = TrackBlock_t::MakeInstanceAt(capacity, buffer2);

  // 2.  Initialise track - on device
  // --------------------------------
  std::cout << " Initialising tracks." << std::endl;
  std::cout << " Max step size = " << maxStepSize << std::endl;

  unsigned  int runId= 101, eventId = 1;
  unsigned  int numTracksEv1 = numTracks / 2;
  initTracks<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, numTracksEv1, eventId, runId );
  initTracks<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, numTracks-numTracksEv1, ++eventId, runId );  
  hipDeviceSynchronize();

  const unsigned int SmallNum= std::max( 2, numTracks);
  // track tracksStart_host[SmallNum];
  
  // hipMemcpy(tracksStart_host, trackBlock_uniq, SmallNum*sizeof(SimpleTrack), hipMemcpyDeviceToHost );

  std::cout << std::endl;
  std::cout << " Initialised tracks: " << std::endl;
  reportTracks( trackBlock_uniq, numTracks );  

  // 3.  Move tracks in field - for one step
  // ----------------------------------------
  std::cout << " Calling move in field (host)." << std::endl;
  for( int i = 0; i<SmallNum ; i++){
     // (*block)[particle_index].energy = energy;     
     track& aTrack = (*trackBlock_uniq)[i];
     // moveInField( aTrack );

     track  ghostTrack = aTrack;
     // reportOneTrack( ghostTrack, i );
     
     moveInField( ghostTrack );
     
     // std::cout << " Track " << i << " addr = " << &aTrack << std::endl;
     // std::cout << " Track " << i << " pdg = " << aTrack.pdg
     //          << " x,y,z = " << aTrack.position[0] << " , " << aTrack.position[1]
     //          << " , " << aTrack.position[3] << std::endl;
     reportOneTrack( ghostTrack, i );   
  }
  // std::cout << " Tracks moved in host: " << std::endl;
  // reportTracks( trackBlock_uniq, numTracks );

  std::cout << std::endl;
  std::cout << " Calling move in field (device)" << std::endl;

  int maxIndex = trackBlock_uniq->GetNused() + trackBlock_uniq->GetNholes();     
  std::cout  << " maxIndex = " << maxIndex
             << " numTracks = " << numTracks << std::endl;
  
  moveInField_glob<<<numBlocks, numThreadsPerBlock>>>(trackBlock_uniq, numTracks);
  //*********
  hipDeviceSynchronize();  

  // 4.  Report result of movement
  // 
  //          See where they went ?
  std::cout << " Ending tracks: " << std::endl;
  reportTracks( trackBlock_uniq, numTracks );
}


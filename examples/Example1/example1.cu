#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2020 CERN
// SPDX-License-Identifier: Apache-2.0

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>

#include <AdePT/BlockData.h>

#include "process.h"
#include "process_list.h"
#include "pair_production.h"
#include "energy_loss.h"

#include "track.h"

#include <AdePT/MParray.h>

// some simple scoring
struct Scoring {
  adept::Atomic_t<int> secondaries;
  adept::Atomic_t<float> totalEnergyLoss;

  VECCORE_ATT_HOST_DEVICE
  Scoring() {}

  VECCORE_ATT_HOST_DEVICE
  static Scoring *MakeInstanceAt(void *addr)
  {
    Scoring *obj = new (addr) Scoring();
    return obj;
  }
};

// kernel select processes based on interaction lenght and put particles in the appropriate queues
__global__ void DefinePhysicalStepLength(adept::BlockData<track> *block, process_list** proclist, adept::MParray **queues, hiprandState_t *states)
{
  int n = block->GetNused() + block->GetNholes();

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    
    // skip particles that are already dead
    if ((*block)[i].status == dead) continue;

    (*proclist)->GetPhysicsInteractionLength(i, block, states); // return value (if step limited by physics or geometry) not used for the moment
    // now, I know which process wins, so I add the particle to the appropriate queue
    queues[(*block)[i].current_process]->push_back(i);
  }
}

// kernel to call Along Step function for particles in the queues
__global__ void CallAlongStepProcesses(adept::BlockData<track> *block, process_list** proclist, adept::MParray **queues, 
                                        Scoring *scor, hiprandState_t *states)
{
  int particle_index;

  // loop over all processes
  for (int process_id=0 ; process_id < (*proclist)->list_size; process_id++) 
    {
      // for each process [process_id] consume the associated queue of particles
      int queue_size = queues[process_id]->size();

      for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < queue_size; i += blockDim.x * gridDim.x) 
        {
          // get particles index from the queue
          particle_index = (*(queues[process_id]))[i];
          // and call the process for it
          ((*proclist)->list)[process_id]->GenerateInteraction(particle_index, block, states);

          // a simple version of scoring 
          scor->totalEnergyLoss.fetch_add((*block)[particle_index].energy_loss);
          scor->secondaries.fetch_add((*block)[particle_index].number_of_secondaries);

          // if particles returns with 'dead' status, release the element from the block
          if ((*block)[particle_index].status == dead) block->ReleaseElement(particle_index);
        }
    }
}

// kernel function to initialize the random states
__global__ void init(hiprandState_t *states)
{
  /* we have to initialize the state */
  hiprand_init(0, 0, 0, states);
}

// kernel to create the processes and process list
__global__ void create_processes(process_list **proclist, process **processes)
{
  // instantiate the existing processes
  *(processes) = new energy_loss();
  *(processes+1) = new pair_production();

  // add them to process_list (process manager)
  *proclist = new process_list(processes, 2);
}

//
int main()
{
  // call the kernel to initialize RND engine
  hiprandState_t *state;
  hipMalloc((void **)&state, sizeof(hiprandState_t));
  init<<<1, 1>>>(state);
  hipDeviceSynchronize();

  // call the kernel to create the processes to be run on the device
  process_list **proclist;
  process **processes;
  hipMalloc((void**)&proclist, sizeof(process_list*));
  hipMalloc((void**)&processes, 2*sizeof(process*));
  create_processes<<<1,1>>>(proclist, processes);
  hipDeviceSynchronize();

  // Capacity of the different containers
  constexpr int capacity = 1 << 20;

  // setting the number of existing processes
  constexpr int numberOfProcesses = 2;
  char *buffer1[numberOfProcesses];

  // reserving queues for each of the processes
  adept::MParray **queues = nullptr;
  hipMallocManaged(&queues, numberOfProcesses * sizeof(adept::MParray *));
  size_t buffersize = adept::MParray::SizeOfInstance(capacity);

  for (int i = 0; i < numberOfProcesses; i++) {
    buffer1[i] = nullptr;
    hipMallocManaged(&buffer1[i], buffersize);
    queues[i] = adept::MParray::MakeInstanceAt(capacity, buffer1[i]);
  }

  // Allocate the content of Scoring in a buffer
  char *buffer_scor = nullptr;
  hipMallocManaged(&buffer_scor, sizeof(Scoring));
  Scoring *scor = Scoring::MakeInstanceAt(buffer_scor);
  // Initialize scoring
  scor->secondaries     = 0;
  scor->totalEnergyLoss = 0;
  
  // Allocate a block of tracks with capacity larger than the total number of spawned threads
  size_t blocksize = adept::BlockData<track>::SizeOfInstance(capacity);
  char *buffer2    = nullptr;
  hipMallocManaged(&buffer2, blocksize);
  auto block = adept::BlockData<track>::MakeInstanceAt(capacity, buffer2);

  // initializing one track in the block
  auto track    = block->NextElement();
  track->energy = 100.0f;
  track->energy_loss = 0.0f;
  //  track->index = 1; // this is not use for the moment, but it should be a unique track index

  // initializing second track in the block
  auto track2    = block->NextElement();
  track2->energy = 30.0f;
  track2->energy_loss = 0.0f;
  //  track2->index = 2; // this is not use for the moment, but it should be a unique track index

  // simple version of scoring
  float* energy_deposition = nullptr;
  hipMalloc((void **)&energy_deposition, sizeof(float));

  constexpr dim3 nthreads(32);
  constexpr dim3 maxBlocks(10);
  dim3 numBlocks;

  while (block->GetNused()>0) 
  {
    numBlocks.x = (block->GetNused() + block->GetNholes() + nthreads.x - 1) / nthreads.x;
    numBlocks.x = std::min(numBlocks.x, maxBlocks.x);

    // call the kernel to do check the step lenght and select process
    DefinePhysicalStepLength<<<numBlocks, nthreads>>>(block, proclist, queues, state);
    
    // call the kernel for Along Step Processes
    CallAlongStepProcesses<<<numBlocks, nthreads>>>(block, proclist, queues, scor, state);

    hipDeviceSynchronize();
    // clear all the queues before next step
    for (int i = 0; i < numberOfProcesses; i++) queues[i]->clear();
    hipDeviceSynchronize();

    std::cout << "Number of tracks in flight: " << std::setw(8) << block->GetNused() << " total energy depostion: " << std::setw(10) << scor->totalEnergyLoss.load() 
    << " total number of secondaries: " << scor->secondaries.load() << std::endl;
  }
}

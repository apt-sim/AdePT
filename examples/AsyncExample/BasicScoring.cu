#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2024 CERN
// SPDX-License-Identifier: Apache-2.0

#include "BasicScoring.h"
// #include "AdeptIntegration.h"

#include <AdePT/copcore/Global.h>

#include <cub/device/device_merge_sort.cuh>

#include <chrono>
#include <mutex>
#include <thread>

// Comparison for sorting tracks into events on device:
struct CompareGPUHits {
  __device__ bool operator()(const GPUHit &lhs, const GPUHit &rhs) const { return lhs.fEventId < rhs.fEventId; }
};

namespace AsyncAdePT {

__device__ HitScoringBuffer gHitScoringBuffer_dev;

__device__ GPUHit &HitScoringBuffer::GetNextSlot()
{
  const auto slotIndex = atomicAdd(&fSlotCounter, 1);
  if (slotIndex >= fNSlot) {
    printf("Trying to score hit #%d with only %d slots\n", slotIndex, fNSlot);
    COPCORE_EXCEPTION("Out of slots in HitScoringBuffer::NextSlot");
  }

  return hitBuffer_dev[slotIndex];
}

HitScoring::HitScoring(unsigned int hitCapacity, unsigned int nThread) : fHitCapacity{hitCapacity}, fHitQueues(nThread)
{
  // We use a single allocation for both buffers:
  GPUHit *gpuHits = nullptr;
  COPCORE_CUDA_CHECK(hipHostMalloc(&gpuHits, sizeof(GPUHit) * 2 * fHitCapacity));
  fGPUHitBuffer_host.reset(gpuHits);

  auto result = hipMalloc(&gpuHits, sizeof(GPUHit) * 2 * fHitCapacity);
  if (result != hipSuccess) throw std::invalid_argument{"No space to allocate hit buffer."};
  fGPUHitBuffer_dev.reset(gpuHits);

  // Init buffers for on-device sorting of hits:
  // Determine device storage requirements for on-device sorting.
  result = hipcub::DeviceMergeSort::SortKeys(nullptr, fGPUSortAuxMemorySize, fGPUHitBuffer_dev.get(), fHitCapacity,
                                          CompareGPUHits{});
  if (result != hipSuccess) throw std::invalid_argument{"No space for hit sorting on device."};

  std::byte *gpuSortingMem;
  result = hipMalloc(&gpuSortingMem, fGPUSortAuxMemorySize);
  if (result != hipSuccess) throw std::invalid_argument{"No space to allocate hit sorting buffer."};
  fGPUSortAuxMemory.reset(gpuSortingMem);

  // Store buffer data in structs
  fBuffers[0].hitScoringInfo = HitScoringBuffer{fGPUHitBuffer_dev.get(), 0, fHitCapacity};
  fBuffers[0].hostBuffer = fGPUHitBuffer_host.get();
  fBuffers[0].state      = BufferHandle::State::OnDevice;
  fBuffers[1].hitScoringInfo = HitScoringBuffer{fGPUHitBuffer_dev.get() + fHitCapacity, 0, fHitCapacity};
  fBuffers[1].hostBuffer = fGPUHitBuffer_host.get() + fHitCapacity;
  fBuffers[1].state      = BufferHandle::State::Free;

  COPCORE_CUDA_CHECK(hipGetSymbolAddress(&fHitScoringBuffer_deviceAddress, HIP_SYMBOL(gHitScoringBuffer_dev)));
  assert(fHitScoringBuffer_deviceAddress != nullptr);
  COPCORE_CUDA_CHECK(hipMemcpy(fHitScoringBuffer_deviceAddress, &fBuffers[0].hitScoringInfo, sizeof(HitScoringBuffer),
                                hipMemcpyHostToDevice));
}

/// Place a new empty buffer on the GPU.
/// The caller has to ensure that all scoring work on the device completes by making the cuda
/// stream wait for all transport on the device.
/// The function will block while the swap is running.
void HitScoring::SwapDeviceBuffers(hipStream_t cudaStream)
{
  // Ensure that host side has been processed:
  auto &currentBuffer = fBuffers[fActiveBuffer];
  if (currentBuffer.state != BufferHandle::State::OnDevice)
    throw std::logic_error(__FILE__ + std::to_string(__LINE__) + ": On-device buffer in wrong state");

  // Get new buffer info from device:
  auto &currentHitInfo = currentBuffer.hitScoringInfo;
  COPCORE_CUDA_CHECK(hipMemcpyAsync(&currentHitInfo, fHitScoringBuffer_deviceAddress, sizeof(HitScoringBuffer),
                                     hipMemcpyDefault, cudaStream));

  // Execute the swap:
  fActiveBuffer          = (fActiveBuffer + 1) % fBuffers.size();
  auto &nextDeviceBuffer = fBuffers[fActiveBuffer];
  while (nextDeviceBuffer.state != BufferHandle::State::Free) {
    std::cerr << __func__ << " Warning: Another thread should have processed the hits.\n";
  }
  assert(nextDeviceBuffer.state == BufferHandle::State::Free && nextDeviceBuffer.hitScoringInfo.fSlotCounter == 0);

  nextDeviceBuffer.state = BufferHandle::State::OnDevice;
  COPCORE_CUDA_CHECK(hipMemcpyAsync(fHitScoringBuffer_deviceAddress, &nextDeviceBuffer.hitScoringInfo,
                                     sizeof(HitScoringBuffer), hipMemcpyDefault, cudaStream));
  COPCORE_CUDA_CHECK(hipStreamSynchronize(cudaStream));
  currentBuffer.state = BufferHandle::State::OnDeviceNeedTransferToHost;
}

/// Copy the current contents of the GPU hit buffer to host.
void HitScoring::TransferHitsToHost(hipStream_t cudaStreamForHitCopy)
{
  for (auto &buffer : fBuffers) {
    if (buffer.state != BufferHandle::State::OnDeviceNeedTransferToHost) continue;

    buffer.state = BufferHandle::State::TransferToHost;
    assert(buffer.hitScoringInfo.fSlotCounter < fHitCapacity);

    auto bufferBegin = buffer.hitScoringInfo.hitBuffer_dev;

    hipcub::DeviceMergeSort::SortKeys(fGPUSortAuxMemory.get(), fGPUSortAuxMemorySize, bufferBegin,
                                   buffer.hitScoringInfo.fSlotCounter, CompareGPUHits{}, cudaStreamForHitCopy);

    COPCORE_CUDA_CHECK(hipMemcpyAsync(buffer.hostBuffer, bufferBegin,
                                       sizeof(GPUHit) * buffer.hitScoringInfo.fSlotCounter, hipMemcpyDefault,
                                       cudaStreamForHitCopy));
    COPCORE_CUDA_CHECK(hipLaunchHostFunc(
        cudaStreamForHitCopy,
        [](void *arg) { static_cast<BufferHandle *>(arg)->state = BufferHandle::State::NeedHostProcessing; }, &buffer));
  }
}

bool HitScoring::ProcessHits()
{
  std::unique_lock lock{fProcessingHitsMutex, std::defer_lock};
  bool haveNewHits = false;

  while (std::any_of(fBuffers.begin(), fBuffers.end(),
                     [](auto &buffer) { return buffer.state >= BufferHandle::State::TransferToHost; })) {
    for (auto &handle : fBuffers) {
      if (handle.state == BufferHandle::State::NeedHostProcessing) {
        if (!lock) lock.lock();
        haveNewHits = true;
        ProcessBuffer(handle);
      }
    }
  }

  return haveNewHits;
}

void HitScoring::ProcessBuffer(BufferHandle &handle)
{
  // We are assuming that the caller holds a lock on fProcessingHitsMutex.
  if (handle.state == BufferHandle::State::NeedHostProcessing) {
    auto hitVector = std::make_shared<std::vector<GPUHit>>();
    hitVector->assign(handle.hostBuffer, handle.hostBuffer + handle.hitScoringInfo.fSlotCounter);
    handle.hitScoringInfo.fSlotCounter = 0;
    handle.state                   = BufferHandle::State::Free;

    for (auto &hitQueue : fHitQueues) {
      hitQueue.push_back(hitVector);
    }
  }
}

std::shared_ptr<const std::vector<GPUHit>> HitScoring::GetNextHitsVector(unsigned int threadId)
{
  assert(threadId < fHitQueues.size());
  std::shared_lock lock{fProcessingHitsMutex};

  if (fHitQueues[threadId].empty())
    return nullptr;
  else {
    auto ret = fHitQueues[threadId].front();
    fHitQueues[threadId].pop_front();
    return ret;
  }
}

/// Clear the device hits content
void PerEventScoring::ClearGPU(hipStream_t cudaStream)
{
  COPCORE_CUDA_CHECK(hipMemsetAsync(fScoring_dev, 0, sizeof(GlobalCounters), cudaStream));
  COPCORE_CUDA_CHECK(hipStreamSynchronize(cudaStream));
}

/// Transfer scoring counters into host instance. Blocks until the operation completes.
void PerEventScoring::CopyToHost(hipStream_t cudaStream)
{
  const auto oldPointer = fScoring_dev;
  COPCORE_CUDA_CHECK(
      hipMemcpyAsync(&fGlobalCounters, fScoring_dev, sizeof(GlobalCounters), hipMemcpyDeviceToHost, cudaStream));
  COPCORE_CUDA_CHECK(hipStreamSynchronize(cudaStream));
  assert(oldPointer == fScoring_dev);
  (void)oldPointer;
}

} // namespace AsyncAdePT

namespace {
/// @brief Utility function to copy a 3D vector, used for filling the Step Points
__device__ __forceinline__ void Copy3DVector(vecgeom::Vector3D<Precision> const &source,
                                             vecgeom::Vector3D<Precision> &destination)
{
  destination = source;
}
} // namespace

namespace adept_scoring {

/// @brief Record a hit
template <>
__device__ void RecordHit(AsyncAdePT::PerEventScoring * /*scoring*/, int aParentID, char aParticleType,
                          double aStepLength, double aTotalEnergyDeposit, vecgeom::NavigationState const *aPreState,
                          vecgeom::Vector3D<Precision> const *aPrePosition,
                          vecgeom::Vector3D<Precision> const *aPreMomentumDirection,
                          vecgeom::Vector3D<Precision> const * /*aPrePolarization*/, double aPreEKin, double aPreCharge,
                          vecgeom::NavigationState const *aPostState, vecgeom::Vector3D<Precision> const *aPostPosition,
                          vecgeom::Vector3D<Precision> const *aPostMomentumDirection,
                          vecgeom::Vector3D<Precision> const * /*aPostPolarization*/, double aPostEKin,
                          double aPostCharge, unsigned int eventID, short threadID)
{
  // Acquire a hit slot
  GPUHit &aGPUHit  = AsyncAdePT::gHitScoringBuffer_dev.GetNextSlot();
  aGPUHit.fParentID = aParentID;
  aGPUHit.fEventId = eventID;
  aGPUHit.threadId = threadID;

  // Fill the required data
  aGPUHit.fParticleType       = aParticleType;
  aGPUHit.fStepLength         = aStepLength;
  aGPUHit.fTotalEnergyDeposit = aTotalEnergyDeposit;
  // Pre step point
  aGPUHit.fPreStepPoint.fNavigationState = *aPreState;
  Copy3DVector(*aPrePosition, aGPUHit.fPreStepPoint.fPosition);
  Copy3DVector(*aPreMomentumDirection, aGPUHit.fPreStepPoint.fMomentumDirection);
  // Copy3DVector(aPrePolarization, aGPUHit.fPreStepPoint.fPolarization);
  aGPUHit.fPreStepPoint.fEKin   = aPreEKin;
  aGPUHit.fPreStepPoint.fCharge = aPreCharge;
  // Post step point
  aGPUHit.fPostStepPoint.fNavigationState = *aPostState;
  Copy3DVector(*aPostPosition, aGPUHit.fPostStepPoint.fPosition);
  Copy3DVector(*aPostMomentumDirection, aGPUHit.fPostStepPoint.fMomentumDirection);
  // Copy3DVector(aPostPolarization, aGPUHit.fPostStepPoint.fPolarization);
  aGPUHit.fPostStepPoint.fEKin   = aPostEKin;
  aGPUHit.fPostStepPoint.fCharge = aPostCharge;
}

template <>
__device__ void AccountProduced(AsyncAdePT::PerEventScoring *scoring, int num_ele, int num_pos, int num_gam)
{
  atomicAdd(&scoring->fGlobalCounters.numElectrons, num_ele);
  atomicAdd(&scoring->fGlobalCounters.numPositrons, num_pos);
  atomicAdd(&scoring->fGlobalCounters.numGammas, num_gam);
}
}
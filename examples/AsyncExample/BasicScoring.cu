#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2022 CERN
// SPDX-License-Identifier: Apache-2.0

#include "BasicScoring.h"
#include "AdeptIntegration.h"

#include <CopCore/Global.h>
#include <CopCore/PhysicalConstants.h>

#include "Track.cuh" // not nice - we expose the track model here, interface of DepositEnergy to be changed

#include <iostream>
#include <iomanip>
#include <stdio.h>

void BasicScoring::InitializeOnGPU(BasicScoring *const BasicScoring_dev)
{
  fAuxData_dev = AdeptIntegration::VolAuxArray::GetInstance().fAuxData_dev;
  // Allocate memory to score charged track length and energy deposit per volume.
  COPCORE_CUDA_CHECK(hipMalloc(&fChargedTrackLength_dev, sizeof(double) * fNumSensitive));
  COPCORE_CUDA_CHECK(hipMalloc(&fEnergyDeposit_dev, sizeof(double) * fNumSensitive));

  // Allocate and initialize scoring and statistics.
  COPCORE_CUDA_CHECK(hipMalloc(&fGlobalScoring_dev, sizeof(GlobalScoring)));

  ScoringPerVolume scoringPerVolume_devPtrs;
  scoringPerVolume_devPtrs.chargedTrackLength = fChargedTrackLength_dev;
  scoringPerVolume_devPtrs.energyDeposit      = fEnergyDeposit_dev;
  COPCORE_CUDA_CHECK(hipMalloc(&fScoringPerVolume_dev, sizeof(ScoringPerVolume)));
  COPCORE_CUDA_CHECK(
      hipMemcpy(fScoringPerVolume_dev, &scoringPerVolume_devPtrs, sizeof(ScoringPerVolume), hipMemcpyHostToDevice));

  // Now copy host instance to device
  COPCORE_CUDA_CHECK(hipMemcpy(BasicScoring_dev, this, sizeof(BasicScoring), hipMemcpyHostToDevice));

  ClearGPU();
}

BasicScoring::BasicScoring(int numSensitive)
    : fNumSensitive{numSensitive}, fChargedTrackLength{new double[numSensitive]},
      fEnergyDeposit{new double[numSensitive]}, fScoringPerVolume{fEnergyDeposit, fChargedTrackLength}
{
}

/// @brief Copy the host side of the scoring. The device side will remain uninitialised.
BasicScoring::BasicScoring(const BasicScoring &other)
    : fNumSensitive{other.fNumSensitive}, fChargedTrackLength{new double[fNumSensitive]},
      fEnergyDeposit{new double[fNumSensitive]}, fScoringPerVolume{fEnergyDeposit, fChargedTrackLength},
      fGlobalScoring{other.fGlobalScoring}
{
  std::copy(other.fChargedTrackLength, other.fChargedTrackLength + fNumSensitive, fChargedTrackLength);
  std::copy(other.fEnergyDeposit, other.fEnergyDeposit + fNumSensitive, fEnergyDeposit);
  // Only one instance can own the device pointers:
  fEnergyDeposit_dev      = nullptr;
  fChargedTrackLength_dev = nullptr;
  fScoringPerVolume_dev   = nullptr;
  fGlobalScoring_dev      = nullptr;
}

/// @brief Move the scoring. The moved-to instance will own the device pointers.
BasicScoring::BasicScoring(BasicScoring &&other)
    : fNumSensitive{other.fNumSensitive}, fAuxData_dev{other.fAuxData_dev},
      fEnergyDeposit_dev{other.fEnergyDeposit_dev}, fChargedTrackLength_dev{other.fChargedTrackLength_dev},
      fScoringPerVolume_dev{other.fScoringPerVolume_dev}, fGlobalScoring_dev{other.fGlobalScoring_dev},
      fChargedTrackLength{std::move(other.fChargedTrackLength)}, fEnergyDeposit{std::move(other.fEnergyDeposit)},
      fScoringPerVolume{std::move(other.fScoringPerVolume)}, fGlobalScoring{std::move(other.fGlobalScoring)}
{
  // Only one instance can own the device pointers:
  other.fEnergyDeposit_dev      = nullptr;
  other.fChargedTrackLength_dev = nullptr;
  other.fScoringPerVolume_dev   = nullptr;
  other.fGlobalScoring_dev      = nullptr;
  other.fChargedTrackLength     = nullptr;
  other.fEnergyDeposit          = nullptr;
}

BasicScoring::~BasicScoring()
{
  FreeGPU();
  delete[] fChargedTrackLength;
  delete[] fEnergyDeposit;
}

void BasicScoring::FreeGPU()
{
  // Free resources.
  COPCORE_CUDA_CHECK(hipFree(fChargedTrackLength_dev));
  COPCORE_CUDA_CHECK(hipFree(fEnergyDeposit_dev));

  COPCORE_CUDA_CHECK(hipFree(fGlobalScoring_dev));
  COPCORE_CUDA_CHECK(hipFree(fScoringPerVolume_dev));
}

void BasicScoring::ClearGPU()
{
  // Clear the device hits content
  COPCORE_CUDA_CHECK(hipMemset(fGlobalScoring_dev, 0, sizeof(GlobalScoring)));
  COPCORE_CUDA_CHECK(hipMemset(fChargedTrackLength_dev, 0, sizeof(double) * fNumSensitive));
  COPCORE_CUDA_CHECK(hipMemset(fEnergyDeposit_dev, 0, sizeof(double) * fNumSensitive));
}

void BasicScoring::CopyHitsToHost()
{
  // Transfer back scoring.
  COPCORE_CUDA_CHECK(hipMemcpy(&fGlobalScoring, fGlobalScoring_dev, sizeof(GlobalScoring), hipMemcpyDeviceToHost));

  // Transfer back the scoring per volume (charged track length and energy deposit).
  COPCORE_CUDA_CHECK(hipMemcpy(fScoringPerVolume.chargedTrackLength, fChargedTrackLength_dev,
                                sizeof(double) * fNumSensitive, hipMemcpyDeviceToHost));
  COPCORE_CUDA_CHECK(hipMemcpy(fScoringPerVolume.energyDeposit, fEnergyDeposit_dev, sizeof(double) * fNumSensitive,
                                hipMemcpyDeviceToHost));
}

__device__ void BasicScoring::Score(vecgeom::NavStateIndex const &crt_state, int charge, double geomStep, double edep)
{
  assert(fGlobalScoring_dev && "Scoring not initialized on device");
  auto volume  = crt_state.Top();
  int volumeID = volume->id();
  int charged  = abs(charge);

  int lvolID = volume->GetLogicalVolume()->id();

  // Add to charged track length, global energy deposit and deposit per volume
  atomicAdd(&fScoringPerVolume_dev->chargedTrackLength[volumeID], charged * geomStep);
  atomicAdd(&fGlobalScoring_dev->energyDeposit, edep);
  atomicAdd(&fScoringPerVolume_dev->energyDeposit[volumeID], edep);
}

__device__ void BasicScoring::AccountHit()
{
  // Increment hit counter
  atomicAdd(&fGlobalScoring_dev->hits, 1);
}

__device__ void BasicScoring::AccountChargedStep(int charge)
{
  // Increase counters for charged/neutral steps
  int charged = abs(charge);
  // Increment global number of steps
  atomicAdd(&fGlobalScoring_dev->chargedSteps, charged);
  atomicAdd(&fGlobalScoring_dev->neutralSteps, 1 - charged);
}

__device__ void BasicScoring::AccountProduced(int num_ele, int num_pos, int num_gam)
{
  // Increment number of secondaries
  atomicAdd(&fGlobalScoring_dev->numElectrons, num_ele);
  atomicAdd(&fGlobalScoring_dev->numPositrons, num_pos);
  atomicAdd(&fGlobalScoring_dev->numGammas, num_gam);
}

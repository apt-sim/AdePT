#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2022 CERN
// SPDX-License-Identifier: Apache-2.0

#include "AdeptIntegration.h"
#include "AdeptIntegration.cuh"

#include <VecGeom/base/Config.h>
#ifdef VECGEOM_ENABLE_CUDA
#include <VecGeom/backend/cuda/Interface.h>
#endif

#include <AdePT/base/Atomic.h>
#include <AdePT/navigation/BVHNavigator.h>
#include <AdePT/base/MParray.h>

#include <AdePT/copcore/Global.h>
#include <AdePT/copcore/PhysicalConstants.h>
#include <AdePT/copcore/Ranluxpp.h>

#include <G4Threading.hh>
#include <G4TransportationManager.hh>
#include <G4UniformMagField.hh>
#include <G4FieldManager.hh>

#include <G4HepEmState.hh>
#include <G4HepEmData.hh>
#include <G4HepEmState.hh>
#include <G4HepEmStateInit.hh>
#include <G4HepEmParameters.hh>
#include <G4HepEmMatCutData.hh>

#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <vector>
#include <numeric>
#include <algorithm>
#include <stdexcept>

#include "electrons.cuh"
#include "gammas.cuh"

#include <AdePT/benchmarking/NVTX.h>

__constant__ __device__ struct G4HepEmParameters g4HepEmPars;
__constant__ __device__ struct G4HepEmData g4HepEmData;

__constant__ __device__ adeptint::VolAuxData *gVolAuxData = nullptr;
__constant__ __device__ double BzFieldValue               = 0;

G4HepEmState *AdeptIntegration::fg4hepem_state{nullptr};

AdeptIntegration::AdeptIntegration(unsigned short nThread, unsigned int trackCapacity, unsigned int bufferThreshold,
                                   int debugLevel, G4Region *region, std::unordered_map<std::string, int> &sensVolIndex,
                                   std::unordered_map<const G4VPhysicalVolume *, int> &scoringMap)
    : fNThread{nThread}, fTrackCapacity{trackCapacity}, fBufferThreshold{bufferThreshold}, fDebugLevel{debugLevel},
      fRegion{region}, sensitive_volume_index{sensVolIndex}, fScoringMap{scoringMap}, fEventStates(nThread),
      fGPUNetEnergy(nThread, 0.)
{
  if (nThread > kMaxThreads)
    throw std::invalid_argument("AdeptIntegration limited to " + std::to_string(kMaxThreads) + " threads");

  for (auto &eventState : fEventStates) {
    std::atomic_init(&eventState, EventState::ScoringRetrieved);
  }

  AdeptIntegration::Initialize();
}

AdeptIntegration::~AdeptIntegration()
{
  FreeGPU();
}

void AdeptIntegration::VolAuxArray::InitializeOnGPU()
{
  // Transfer volume auxiliary data
  COPCORE_CUDA_CHECK(hipMalloc(&fAuxData_dev, sizeof(VolAuxData) * fNumVolumes));
  COPCORE_CUDA_CHECK(hipMemcpy(fAuxData_dev, fAuxData, sizeof(VolAuxData) * fNumVolumes, hipMemcpyHostToDevice));
  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gVolAuxData), &fAuxData_dev, sizeof(VolAuxData *)));
}

void AdeptIntegration::VolAuxArray::FreeGPU()
{
  COPCORE_CUDA_CHECK(hipFree(fAuxData_dev));
}

static G4HepEmState *InitG4HepEm()
{
  auto state = new G4HepEmState;
  InitG4HepEmState(state);

  G4HepEmMatCutData *cutData = state->fData->fTheMatCutData;
  G4cout << "fNumG4MatCuts = " << cutData->fNumG4MatCuts << ", fNumMatCutData = " << cutData->fNumMatCutData << G4endl;

  // Copy to GPU.
  CopyG4HepEmDataToGPU(state->fData);
  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g4HepEmPars), state->fParameters, sizeof(G4HepEmParameters)));

  // Create G4HepEmData with the device pointers.
  G4HepEmData dataOnDevice;
  dataOnDevice.fTheMatCutData   = state->fData->fTheMatCutData_gpu;
  dataOnDevice.fTheMaterialData = state->fData->fTheMaterialData_gpu;
  dataOnDevice.fTheElementData  = state->fData->fTheElementData_gpu;
  dataOnDevice.fTheElectronData = state->fData->fTheElectronData_gpu;
  dataOnDevice.fThePositronData = state->fData->fThePositronData_gpu;
  dataOnDevice.fTheSBTableData  = state->fData->fTheSBTableData_gpu;
  dataOnDevice.fTheGammaData    = state->fData->fTheGammaData_gpu;
  // The other pointers should never be used.
  dataOnDevice.fTheMatCutData_gpu   = nullptr;
  dataOnDevice.fTheMaterialData_gpu = nullptr;
  dataOnDevice.fTheElementData_gpu  = nullptr;
  dataOnDevice.fTheElectronData_gpu = nullptr;
  dataOnDevice.fThePositronData_gpu = nullptr;
  dataOnDevice.fTheSBTableData_gpu  = nullptr;
  dataOnDevice.fTheGammaData_gpu    = nullptr;

  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g4HepEmData), &dataOnDevice, sizeof(G4HepEmData)));

  return state;
}

// Kernel to initialize the set of queues per particle type.
__global__ void InitParticleQueues(ParticleQueues queues, size_t Capacity)
{
  adept::MParray::MakeInstanceAt(Capacity, queues.currentlyActive);
  adept::MParray::MakeInstanceAt(Capacity, queues.nextActive);
  adept::MParray::MakeInstanceAt(Capacity, queues.leakedTracksCurrent);
  adept::MParray::MakeInstanceAt(Capacity, queues.leakedTracksNext);
}

// Init a queue at the designated location
template <typename T>
__global__ void InitQueue(adept::MParrayT<T> *queue, size_t Capacity)
{
  adept::MParrayT<T>::MakeInstanceAt(Capacity, queue);
}

// Kernel function to initialize tracks comming from a Geant4 buffer
__global__ void InjectTracks(adeptint::TrackData *trackinfo, int ntracks, Secondaries secondaries,
                             const vecgeom::VPlacedVolume *world, AdeptScoring *userScoring,
                             adept::MParrayT<QueueIndexPair> *toBeEnqueued)
{
  constexpr double tolerance = 10. * vecgeom::kTolerance;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ntracks; i += blockDim.x * gridDim.x) {
    ParticleGenerator *generator = nullptr;
    const auto &trackInfo        = trackinfo[i];
    short queueIndex             = -1;
    switch (trackInfo.pdg) {
    case 11:
      generator  = &secondaries.electrons;
      queueIndex = ParticleType::Electron;
      break;
    case -11:
      generator  = &secondaries.positrons;
      queueIndex = ParticleType::Positron;
      break;
    case 22:
      generator  = &secondaries.gammas;
      queueIndex = ParticleType::Gamma;
    };
    assert(generator != nullptr && "Unsupported pdg type");

    // TODO: Delay when not enough slots?
    const auto slot = generator->fSlotManager->NextSlot();
#if false
    printf("\t%d,%d: Obtained slot %d for track %d/%d (%d, %d, %d, %d). Slots: (%d %d %d)\n", blockIdx.x, threadIdx.x,
           slot, i, ntracks, trackInfo.eventId, trackInfo.threadId, trackInfo.trackId, trackInfo.pdg,
           secondaries.electrons.fSlotManager->OccupiedSlots(), secondaries.positrons.fSlotManager->OccupiedSlots(),
           secondaries.gammas.fSlotManager->OccupiedSlots());
#endif
    toBeEnqueued->push_back(QueueIndexPair{slot, queueIndex});
    Track &track = generator->fTracks[slot];
    track.rngState.SetSeed(1234567 * trackInfo.eventId + trackInfo.trackId);
    track.energy       = trackInfo.energy;
    track.numIALeft[0] = -1.0;
    track.numIALeft[1] = -1.0;
    track.numIALeft[2] = -1.0;

    track.initialRange       = -1.0;
    track.dynamicRangeFactor = -1.0;
    track.tlimitMin          = -1.0;

    track.pos = {trackInfo.position[0], trackInfo.position[1], trackInfo.position[2]};
    track.dir = {trackInfo.direction[0], trackInfo.direction[1], trackInfo.direction[2]};

    track.eventId  = trackInfo.eventId;
    track.threadId = trackInfo.threadId;

    // We locate the pushed point because we run the risk that the
    // point is not located in the GPU region
#ifdef NDEBUG
    constexpr int maxAttempt = 2;
#else
    constexpr int maxAttempt = 10;
#endif
    for (int attempt = 1; attempt < maxAttempt; ++attempt) {
      const auto amount = attempt < 5 ? attempt : (attempt - 5) * -1;
      track.navState.Clear();
      const auto pushedPosition = track.pos + amount * tolerance * track.dir;
      BVHNavigator::LocatePointIn(world, pushedPosition, track.navState, true);
      // The track must be on boundary at this point
      track.navState.SetBoundaryState(true);
      // nextState is initialized as needed.
      const vecgeom::VPlacedVolume *volume = track.navState.Top();
      int lvolID                           = volume->GetLogicalVolume()->id();
      adeptint::VolAuxData const &auxData  = userScoring[trackInfo.threadId].GetAuxData_dev(lvolID);
#ifndef NDEBUG
      if (auxData.fGPUregion && attempt == 1) {
        break;
      } else {
        printf("Error [%d, %d]: ev=%d track=%d: scoring[tid=%d].GetAux_dev[lvolID=%d].fGPUregion=%d volID=%d "
               "x=(%18.15f, %18.15f, %18.15f) dir=(%f, %f, %f) "
               "Safety=%17.15f DistanceToOut=%f shiftAmount=%d\n",
               blockIdx.x, threadIdx.x, trackInfo.eventId, trackInfo.trackId, trackInfo.threadId, lvolID,
               auxData.fGPUregion, volume->id(), pushedPosition[0], pushedPosition[1], pushedPosition[2], track.dir[0],
               track.dir[1], track.dir[2], BVHNavigator::ComputeSafety(pushedPosition, track.navState),
               volume->DistanceToOut(track.pos, track.dir), amount);
        track.navState.Print();
        if (auxData.fGPUregion) {
          printf("Success in attempt %d shiftAmount %d\n", attempt, amount);
          break;
        }
      }
#endif
    }
  }
}

__global__ void EnqueueTracks(AllParticleQueues allQueues, adept::MParrayT<QueueIndexPair> *toBeEnqueued)
{
  const auto end = toBeEnqueued->size();
  for (unsigned int i = threadIdx.x; i < end; i += blockDim.x) {
    const auto [slotNumber, particleType] = (*toBeEnqueued)[i];
    allQueues.queues[particleType].nextActive->push_back(slotNumber);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    toBeEnqueued->clear();
  }
}

__device__ unsigned int nFromDevice_dev;

// Copy particles leaked from the GPU region into a compact buffer
__global__ void FillFromDeviceBuffer(AllLeaked all, adeptint::TrackData *fromDevice, unsigned int maxFromDeviceBuffer)
{
  const auto numElectrons = all.leakedElectrons.fLeakedQueue->size();
  const auto numPositrons = all.leakedPositrons.fLeakedQueue->size();
  const auto numGammas    = all.leakedGammas.fLeakedQueue->size();
  const auto total        = numElectrons + numPositrons + numGammas;
  if (blockIdx.x == 0 && threadIdx.x == 0) nFromDevice_dev = total < maxFromDeviceBuffer ? total : maxFromDeviceBuffer;

  for (unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; i < total; i += blockDim.x * gridDim.x) {
    LeakedTracks *leakedTracks = nullptr;
    unsigned int queueSlot     = 0;
    int pdg                    = 0;

    if (i < numGammas) {
      leakedTracks = &all.leakedGammas;
      queueSlot    = i;
      pdg          = 22;
    } else if (i < numGammas + numElectrons) {
      leakedTracks = &all.leakedElectrons;
      queueSlot    = i - numGammas;
      pdg          = 11;
    } else {
      leakedTracks = &all.leakedPositrons;
      queueSlot    = i - numGammas - numElectrons;
      pdg          = -11;
    }

    const auto trackSlot = (*leakedTracks->fLeakedQueue)[queueSlot];
    Track const *const track = leakedTracks->fTracks + trackSlot;

    if (i >= maxFromDeviceBuffer) {
      // No space to transfer it out
      leakedTracks->fLeakedQueueNext->push_back(trackSlot);
    } else {
      fromDevice[i].position[0]  = track->pos[0];
      fromDevice[i].position[1]  = track->pos[1];
      fromDevice[i].position[2]  = track->pos[2];
      fromDevice[i].direction[0] = track->dir[0];
      fromDevice[i].direction[1] = track->dir[1];
      fromDevice[i].direction[2] = track->dir[2];
      fromDevice[i].energy       = track->energy;
      fromDevice[i].pdg          = pdg;
      fromDevice[i].threadId     = track->threadId;
      fromDevice[i].eventId      = track->eventId;

      leakedTracks->fSlotManager->MarkSlotForFreeing(trackSlot);
    }
  }
}

__global__ void FreeSlots(TracksAndSlots tracksAndSlots)
{
  constexpr auto nSlotMgrs = sizeof(tracksAndSlots.slotManagers) / sizeof(tracksAndSlots.slotManagers[0]);
  for (unsigned int i = blockIdx.x; i < nSlotMgrs; i += gridDim.x) {
    tracksAndSlots.slotManagers[i]->FreeMarkedSlots();
  }
}

// Finish iteration: clear queues and fill statistics.
__global__ void FinishIteration(AllParticleQueues all, Stats *stats, TracksAndSlots tracksAndSlots,
                                GammaInteractions gammaInteractions)
{
  if (blockIdx.x == 0) {
    // Clear queues and write statistics
    for (int i = threadIdx.x; i < ParticleType::NumParticleTypes; i += blockDim.x) {
      all.queues[i].currentlyActive->clear();
      stats->inFlight[i]     = all.queues[i].nextActive->size();
      stats->leakedTracks[i] = all.queues[i].leakedTracksCurrent->size() + all.queues[i].leakedTracksNext->size();
      stats->usedSlots[i]    = tracksAndSlots.slotManagers[i]->OccupiedSlots();
    }
  } else if (blockIdx.x == 1) {
    // Assert that there is enough slots allocated:
    for (int i = threadIdx.x; i < ParticleType::NumParticleTypes; i += blockDim.x) {
      if (all.queues[i].nextActive->size() > tracksAndSlots.slotManagers[i]->OccupiedSlots()) {
        printf("Error particle type %d: %ld in flight while %d slots allocated\n", i, all.queues[i].nextActive->size(),
               tracksAndSlots.slotManagers[i]->OccupiedSlots());
        asm("trap;");
      }
    }
  } else if (blockIdx.x == 2) {
    if (threadIdx.x < gammaInteractions.NInt) {
      gammaInteractions.queues[threadIdx.x]->clear();
    }
  }

#if false
  if (blockIdx.x == 3) {
    for (unsigned int i = threadIdx.x; i < 3; ++i) {
      if (tracksAndSlots.slotManagers[i]->OccupiedSlots() < all.queues[i].nextActive->size())
        printf("Error: For particle %d, %d slots are allocated for %ld in flight\n", i,
               tracksAndSlots.slotManagers[i]->OccupiedSlots(), all.queues[i].nextActive->size());
    }

    if (threadIdx.x == 4) {
      printf("In flight (kernel): %ld %ld %ld  %ld\tslots: %d %d %d\n", all.queues[0].nextActive->size(),
             all.queues[1].nextActive->size(), all.queues[2].nextActive->size(),
             all.queues[0].nextActive->size() + all.queues[1].nextActive->size() + all.queues[2].nextActive->size(),
             tracksAndSlots.slotManagers[0]->OccupiedSlots(), tracksAndSlots.slotManagers[1]->OccupiedSlots(),
             tracksAndSlots.slotManagers[2]->OccupiedSlots());
    }
  }
#endif
}

__global__ void ZeroEventCounters(Stats *stats)
{
  constexpr auto size = std::extent<decltype(stats->perEventInFlight)>::value;
  for (unsigned int i = threadIdx.x; i < size; i += blockDim.x) {
    stats->perEventInFlight[i] = 0;
    stats->perEventLeaked[i]   = 0;
  }
}

/**
 * Count how many tracks are currently in flight for each event.
 */
__global__ void CountCurrentPopulation(AllParticleQueues all, Stats *stats, TracksAndSlots tracksAndSlots)
{
  constexpr unsigned int N = AdeptIntegration::kMaxThreads;
  __shared__ unsigned int sharedCount[N];

  for (unsigned int particleType = blockIdx.x; particleType < ParticleType::NumParticleTypes;
       particleType += gridDim.x) {
    Track const *const tracks   = tracksAndSlots.tracks[particleType];
    adept::MParray const *queue = all.queues[particleType].currentlyActive;

    for (unsigned int i = threadIdx.x; i < N; i += blockDim.x)
      sharedCount[i] = 0;

    __syncthreads();

    const auto end = queue->size();
    for (unsigned int i = threadIdx.x; i < end; i += blockDim.x) {
      const auto slot     = (*queue)[i];
      const auto threadId = tracks[slot].threadId;
      atomicAdd(sharedCount + threadId, 1u);
    }

    __syncthreads();

    for (unsigned int i = threadIdx.x; i < N; i += blockDim.x)
      atomicAdd(stats->perEventInFlight + i, sharedCount[i]);

    __syncthreads();
  }
}

/**
 * Count tracks both in the current and the future queue of leaked particles.
 */
__global__ void CountLeakedTracks(AllParticleQueues all, Stats *stats, TracksAndSlots tracksAndSlots)
{
  constexpr auto nQueue = 2 * ParticleType::NumParticleTypes;
  for (unsigned int queueIndex = blockIdx.x; queueIndex < nQueue; queueIndex += gridDim.x) {
    const auto particleType   = queueIndex / 2;
    Track const *const tracks = tracksAndSlots.tracks[particleType];
    auto const queue = queueIndex < ParticleType::NumParticleTypes ? all.queues[particleType].leakedTracksCurrent
                                                                   : all.queues[particleType].leakedTracksNext;
    const auto end   = queue->size();
    for (unsigned int i = threadIdx.x; i < end; i += blockDim.x) {
      const auto slot     = (*queue)[i];
      const auto threadId = tracks[slot].threadId;
      atomicAdd(stats->perEventLeaked + threadId, 1u);
    }
  }
}

template <typename... Args>
__global__ void ClearQueues(Args *...queue)
{
  (queue->clear(), ...);
}

__global__ void ClearAllQueues(AllParticleQueues all)
{
  for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
    all.queues[i].currentlyActive->clear();
    all.queues[i].nextActive->clear();
    all.queues[i].leakedTracksCurrent->clear();
    all.queues[i].leakedTracksNext->clear();
  }
}

__global__ void InitSlotManagers(SlotManager *mgr, std::size_t N)
{
  for (int i = 0; i < N; ++i) {
    mgr[i].Clear();
  }
}

#ifndef NDEBUG
__global__ void AssertConsistencyOfSlotManagers(SlotManager *mgrs, std::size_t N)
{
  for (int i = 0; i < N; ++i) {
    SlotManager &mgr = mgrs[i];
    const auto slotCounter = mgr.fSlotCounter;
    const auto freeCounter = mgr.fFreeCounter;

    if (blockIdx.x == 0 && threadIdx.x == 0 && slotCounter < freeCounter) {
      printf("Error %s:%d: Trying to free %d slots in manager %d whereas only %d allocated\n", __FILE__, __LINE__,
             freeCounter, i, slotCounter);
      for (unsigned int i = 0; i < freeCounter; ++i) {
        printf("%d ", mgr.fToFreeList[i]);
      }
      printf("\n");
      assert(false);
    }

    bool doubleFree = false;
    for (unsigned int j = blockIdx.x; j < mgr.fFreeCounter; j += gridDim.x) {
      const auto slotToSearch = mgr.fToFreeList[j];
      for (unsigned int k = j + 1 + threadIdx.x; k < freeCounter; k += blockDim.x) {
        if (slotToSearch == mgr.fToFreeList[k]) {
          printf("Error: Manager %d: Slot %d freed both at %d and at %d\n", i, slotToSearch, k, j);
          doubleFree = true;
          break;
        }
      }
    }

    assert(slotCounter == mgr.fSlotCounter && "Race condition while checking slots");
    assert(freeCounter == mgr.fFreeCounter && "Race condition while checking slots");
    assert(!doubleFree);
  }
}
#endif

bool AdeptIntegration::InitializeGeometry(const vecgeom::cxx::VPlacedVolume *world)
{
#ifndef NDEBUG
  COPCORE_CUDA_CHECK(vecgeom::cxx::CudaDeviceSetStackLimit(16384 * 2));
#else
  COPCORE_CUDA_CHECK(vecgeom::cxx::CudaDeviceSetStackLimit(16384));
#endif

  // Upload geometry to GPU.
  auto &cudaManager = vecgeom::cxx::CudaManager::Instance();
  cudaManager.LoadGeometry(world);
  auto world_dev = cudaManager.Synchronize();
  // Initialize BVH
  InitBVH();

  return (world_dev != nullptr);
}

bool AdeptIntegration::InitializePhysics()
{
  // Initialize shared physics data
  AdeptIntegration::fg4hepem_state = InitG4HepEm();
  // Initialize field
  double bz = 0;
  auto field =
      (G4UniformMagField *)G4TransportationManager::GetTransportationManager()->GetFieldManager()->GetDetectorField();
  if (field) {
    auto field_vect = field->GetConstantFieldValue();
    bz              = field_vect[2];
  }
  COPCORE_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(BzFieldValue), &bz, sizeof(double)));

  return true;
}

namespace {
void allocToDeviceTrackData(GPUstate &gpuState, unsigned int numToDevice)
{
  using TrackData       = adeptint::TrackData;
  gpuState.fNumToDevice = numToDevice;
  TrackData *devPtr, *hostPtr;
  COPCORE_CUDA_CHECK(
      hipHostMalloc(&hostPtr, 2 * numToDevice * sizeof(TrackData))); // Double the size to switch between buffers
  COPCORE_CUDA_CHECK(hipMalloc(&devPtr, numToDevice * sizeof(TrackData)));
  gpuState.toDevice_host = {hostPtr, adeptint::cudaHostDeleter};
  gpuState.toDevice_dev  = {devPtr, adeptint::cudaDeleter};
}

void allocFromDeviceTrackData(GPUstate &gpuState, unsigned int numFromDevice)
{
  using TrackData         = adeptint::TrackData;
  gpuState.fNumFromDevice = numFromDevice;
  adeptint::TrackData *devPtr, *hostPtr;
  COPCORE_CUDA_CHECK(hipHostMalloc(&hostPtr, numFromDevice * sizeof(TrackData)));
  COPCORE_CUDA_CHECK(hipMalloc(&devPtr, numFromDevice * sizeof(TrackData)));
  gpuState.fromDevice_host = {hostPtr, adeptint::cudaHostDeleter};
  gpuState.fromDevice_dev  = {devPtr, adeptint::cudaDeleter};
}
} // namespace

void AdeptIntegration::InitializeGPU()
{
  using TrackData    = adeptint::TrackData;
  fGPUstate          = std::make_unique<GPUstate>();
  GPUstate &gpuState = *fGPUstate;

  // Allocate structures to manage tracks of an implicit type:
  //  * memory to hold the actual Track elements,
  //  * objects to manage slots inside the memory,
  //  * queues of slots to remember active particle and those needing relocation,
  //  * a stream and an event for synchronization of kernels.
  size_t TracksSize      = sizeof(Track) * fTrackCapacity;
  const size_t QueueSize = adept::MParray::SizeOfInstance(fTrackCapacity);

  auto gpuMalloc = [&gpuState](auto &devPtr, std::size_t N) {
    COPCORE_CUDA_CHECK(hipMalloc(&devPtr, sizeof(*devPtr) * N));
    gpuState.allCudaPointers.emplace_back(devPtr, adeptint::cudaDeleter);
  };

  SlotManager *slotManagers_dev = nullptr;
  gpuMalloc(slotManagers_dev, ParticleType::NumParticleTypes);

  // Create a stream to synchronize kernels of all particle types.
  COPCORE_CUDA_CHECK(hipStreamCreate(&gpuState.stream));
  for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
    ParticleType &particleType = gpuState.particles[i];

    // Share hepem state between threads
    COPCORE_CUDA_CHECK(hipMalloc(&particleType.tracks, TracksSize));

    particleType.slotManager_host = SlotManager{static_cast<SlotManager::value_type>(fTrackCapacity),
                                                static_cast<SlotManager::value_type>(fTrackCapacity)};
    particleType.slotManager      = slotManagers_dev + i;
    COPCORE_CUDA_CHECK(
        hipMemcpy(particleType.slotManager, &particleType.slotManager_host, sizeof(SlotManager), hipMemcpyDefault));

    COPCORE_CUDA_CHECK(hipMalloc(&particleType.queues.currentlyActive, QueueSize));
    COPCORE_CUDA_CHECK(hipMalloc(&particleType.queues.nextActive, QueueSize));
    COPCORE_CUDA_CHECK(hipMalloc(&particleType.queues.leakedTracksCurrent, QueueSize));
    COPCORE_CUDA_CHECK(hipMalloc(&particleType.queues.leakedTracksNext, QueueSize));
    InitParticleQueues<<<1, 1>>>(particleType.queues, fTrackCapacity);

    COPCORE_CUDA_CHECK(hipStreamCreate(&particleType.stream));
    COPCORE_CUDA_CHECK(hipEventCreate(&particleType.event));
  }

  // init gamma interaction queues
  for (unsigned int i = 0; i < GammaInteractions::NInt; ++i) {
    const auto capacity     = fTrackCapacity / 3;
    const auto instanceSize = adept::MParrayT<GammaInteractions::Data>::SizeOfInstance(capacity);
    COPCORE_CUDA_CHECK(hipMalloc(&gpuState.gammaInteractions.queues[i], instanceSize));
    InitQueue<GammaInteractions::Data><<<1, 1>>>(gpuState.gammaInteractions.queues[i], capacity);
  }

  // initialize statistics
  COPCORE_CUDA_CHECK(hipMalloc(&gpuState.stats_dev, sizeof(Stats)));
  COPCORE_CUDA_CHECK(hipHostMalloc(&gpuState.stats, sizeof(Stats)));

  // init scoring on device
  gpuMalloc(fScoring_dev, fScoring.size());
  for (unsigned int i = 0; i < fNThread; ++i) {
    fScoring[i].InitializeOnGPU(fScoring_dev + i);
  }

  // initialize buffers for track transfer on host and device
  allocToDeviceTrackData(gpuState, gpuState.fNumToDevice);
  allocFromDeviceTrackData(gpuState, gpuState.fNumFromDevice);
  unsigned int *nFromDevice_host = nullptr;
  COPCORE_CUDA_CHECK(hipHostMalloc(&nFromDevice_host, sizeof(unsigned int)));
  gpuState.nFromDevice.reset(nFromDevice_host);

  fBuffer = std::make_unique<adeptint::TrackBuffer>(gpuState.toDevice_host.get(), gpuState.fNumToDevice,
                                                    gpuState.toDevice_host.get() + gpuState.fNumToDevice,
                                                    gpuState.fNumToDevice, fNThread);

  const auto injectQueueSize = adept::MParrayT<QueueIndexPair>::SizeOfInstance(gpuState.fNumToDevice);
  adept::MParrayT<QueueIndexPair> *injectQueue;
  COPCORE_CUDA_CHECK(hipMalloc(&injectQueue, injectQueueSize));
  gpuState.injectionQueue = {injectQueue, adeptint::cudaDeleter};
  InitQueue<QueueIndexPair><<<1, 1>>>(gpuState.injectionQueue.get(), gpuState.fNumToDevice);
}

void AdeptIntegration::FreeGPU()
{
  fGPUstate->runTransport = false;
  fGPUWorker.join();

  // Free resources.
  GPUstate &gpuState = const_cast<GPUstate &>(*fGPUstate);
  COPCORE_CUDA_CHECK(hipFree(gpuState.stats_dev));
  COPCORE_CUDA_CHECK(hipHostFree(gpuState.stats));

  COPCORE_CUDA_CHECK(hipStreamDestroy(gpuState.stream));

  for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
    COPCORE_CUDA_CHECK(hipFree(gpuState.particles[i].tracks));

    COPCORE_CUDA_CHECK(hipFree(gpuState.particles[i].queues.currentlyActive));
    COPCORE_CUDA_CHECK(hipFree(gpuState.particles[i].queues.nextActive));
    COPCORE_CUDA_CHECK(hipFree(gpuState.particles[i].queues.leakedTracksCurrent));
    COPCORE_CUDA_CHECK(hipFree(gpuState.particles[i].queues.leakedTracksNext));

    COPCORE_CUDA_CHECK(hipStreamDestroy(gpuState.particles[i].stream));
    COPCORE_CUDA_CHECK(hipEventDestroy(gpuState.particles[i].event));
  }

  for (auto queue : gpuState.gammaInteractions.queues) {
    COPCORE_CUDA_CHECK(hipFree(queue));
  }

  gpuState.allCudaPointers.clear();

  // Free G4HepEm data
  FreeG4HepEmData(AdeptIntegration::fg4hepem_state->fData);
  delete AdeptIntegration::fg4hepem_state;
  AdeptIntegration::fg4hepem_state = nullptr;
}

void AdeptIntegration::ReturnTracksToG4()
{
  using TrackData                      = adeptint::TrackData;
  const auto &fromDevice               = fGPUstate->fromDevice_host.get();
  TrackData const *const fromDeviceEnd = fromDevice + *fGPUstate->nFromDevice;
  std::scoped_lock lock{fBuffer->fromDeviceMutex};

  for (TrackData *trackIt = fromDevice; trackIt < fromDeviceEnd; ++trackIt) {
    assert(0 <= trackIt->threadId && trackIt->threadId <= fNThread);
    fBuffer->fromDeviceBuffers[trackIt->threadId].push_back(*trackIt);
  }

  AdvanceEventStates(EventState::SecondFlush, EventState::DeviceFlushed);
  fGPUstate->extractState = GPUstate::ExtractState::Idle;

#ifndef NDEBUG
  for (const auto &trackBuffer : fBuffer->fromDeviceBuffers) {
    if (trackBuffer.empty()) continue;
    const auto eventId = trackBuffer.front().eventId;
    assert(std::all_of(trackBuffer.begin(), trackBuffer.end(),
                       [eventId](const TrackData &track) { return eventId == track.eventId; }));
  }
#endif
}

void AdeptIntegration::AdvanceEventStates(EventState oldState, EventState newState)
{
  for (auto &eventState : fEventStates) {
    EventState expected = oldState;
    eventState.compare_exchange_strong(expected, newState, std::memory_order_release, std::memory_order_relaxed);
  }
}

void AdeptIntegration::TransportLoop()
{
  NVTXTracer tracer{"TransportLoop"};

  using TrackData = adeptint::TrackData;
  using InjectState  = GPUstate::InjectState;
  using ExtractState = GPUstate::ExtractState;
  // Capacity of the different containers aka the maximum number of particles.
  auto &cudaManager                             = vecgeom::cxx::CudaManager::Instance();
  const vecgeom::cuda::VPlacedVolume *world_dev = cudaManager.world_gpu();
  GPUstate &gpuState                            = *fGPUstate;

  ParticleType &electrons = gpuState.particles[ParticleType::Electron];
  ParticleType &positrons = gpuState.particles[ParticleType::Positron];
  ParticleType &gammas    = gpuState.particles[ParticleType::Gamma];

  hipEvent_t cudaEvent, cudaStatsEvent;
  hipStream_t transferStream, statsStream, interactionStream;
  COPCORE_CUDA_CHECK(hipEventCreateWithFlags(&cudaEvent, hipEventDisableTiming));
  COPCORE_CUDA_CHECK(hipEventCreateWithFlags(&cudaStatsEvent, hipEventDisableTiming));
  adeptint::unique_ptr_cudaEvent cudaEventCleanup{&cudaEvent, adeptint::cudaEventDeleter};
  adeptint::unique_ptr_cudaEvent cudaStatsEventCleanup{&cudaStatsEvent, adeptint::cudaEventDeleter};
  COPCORE_CUDA_CHECK(hipStreamCreate(&transferStream));
  COPCORE_CUDA_CHECK(hipStreamCreate(&statsStream));
  COPCORE_CUDA_CHECK(hipStreamCreate(&interactionStream));
  std::unique_ptr<hipStream_t, decltype(&adeptint::cudaStreamDeleter)> cudaStreamCleanup{&transferStream,
                                                                                          adeptint::cudaStreamDeleter};
  std::unique_ptr<hipStream_t, decltype(&adeptint::cudaStreamDeleter)> cudaStatsStreamCleanup{
      &statsStream, adeptint::cudaStreamDeleter};
  std::unique_ptr<hipStream_t, decltype(&adeptint::cudaStreamDeleter)> cudaInteractionStreamCleanup{
      &interactionStream, adeptint::cudaStreamDeleter};
  auto waitForOtherStream = [&cudaEvent](hipStream_t waitingStream, hipStream_t streamToWaitFor) {
    COPCORE_CUDA_CHECK(hipEventRecord(cudaEvent, streamToWaitFor));
    COPCORE_CUDA_CHECK(hipStreamWaitEvent(waitingStream, cudaEvent));
  };

  auto computeThreadsAndBlocks = [](unsigned int nParticles) -> std::pair<unsigned int, unsigned int> {
    constexpr int TransportThreads             = 256;
    constexpr int LowOccupancyTransportThreads = 32;

    auto transportBlocks = nParticles / TransportThreads + 1;
    if (transportBlocks < 10) {
      transportBlocks = nParticles / LowOccupancyTransportThreads + 1;
      return {LowOccupancyTransportThreads, transportBlocks};
    }
    return {TransportThreads, transportBlocks};
  };

  SlotManager *const slotMgrArray = gpuState.particles[0].slotManager;
  while (gpuState.runTransport) {
    NVTXTracer nvtx1{"Setup"}, nvtx2{"Setup2"};
    InitSlotManagers<<<80, 256, 0, gpuState.stream>>>(slotMgrArray, ParticleType::NumParticleTypes);
    COPCORE_CUDA_CHECK(hipMemsetAsync(gpuState.stats_dev, 0, sizeof(Stats), gpuState.stream));

    int inFlight                                                   = 0;
    unsigned int numLeaked                                         = 0;
    unsigned int particlesInFlight[ParticleType::NumParticleTypes] = {1, 1, 1};
    int loopingNo                                                  = 0;
    int previousElectrons = -1, previousPositrons = -1;

    auto needTransport = [](std::atomic<EventState> const &state) {
      return state.load(std::memory_order_acquire) < EventState::LeakedTracksRetrieved;
    };
    // Wait for work from G4 workers:
    while (gpuState.runTransport && std::none_of(fEventStates.begin(), fEventStates.end(), needTransport)) {
      using namespace std::chrono_literals;
      std::this_thread::sleep_for(10ms);
    }

    if (fDebugLevel > 2) {
      G4cout << "GPU transport starting" << std::endl;
    }

    COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));

#ifdef USE_NVTX
    std::map<AdeptIntegration::EventState, std::string> stateMap{
        {EventState::NewTracksFromG4, "NewTracksFromG4"},
        {EventState::G4RequestsFlush, "G4RequestsFlush"},
        {EventState::Inject, "Inject"},
        {EventState::InjectionCompleted, "InjectionCompleted"},
        {EventState::Transporting, "Transporting"},
        {EventState::WaitingForTransportToFinish, "WaitingForTransportToFinish"},
        {EventState::NeedDeviceFlush, "NeedDeviceFlush"},
        {EventState::FirstFlush, "FirstFlush"},
        {EventState::SecondFlush, "SecondFlush"},
        {EventState::DeviceFlushed, "DeviceFlushed"},
        {EventState::LeakedTracksRetrieved, "LeakedTracksRetrieved"},
        {EventState::ScoringRetrieved, "ScoringRetrieved"}};
#endif

    for (unsigned int iteration = 0;
         inFlight > 0 || gpuState.injectState != InjectState::Idle || gpuState.extractState != ExtractState::Idle ||
         std::any_of(fEventStates.begin(), fEventStates.end(), needTransport);
         ++iteration) {
#ifdef USE_NVTX
      nvtx1.setTag(stateMap[fEventStates[0].load()].data());
      nvtx2.setTag(stateMap[fEventStates[1].load()].data());
#endif

      // Swap the queues for the next iteration.
      electrons.queues.SwapActive();
      positrons.queues.SwapActive();
      gammas.queues.SwapActive();

      const Secondaries secondaries = {
          .electrons = {electrons.tracks, electrons.slotManager, electrons.queues.nextActive},
          .positrons = {positrons.tracks, positrons.slotManager, positrons.queues.nextActive},
          .gammas    = {gammas.tracks, gammas.slotManager, gammas.queues.nextActive},
      };
      const AllParticleQueues allParticleQueues = {{electrons.queues, positrons.queues, gammas.queues}};
      const TracksAndSlots tracksAndSlots       = {{electrons.tracks, positrons.tracks, gammas.tracks},
                                                   {electrons.slotManager, positrons.slotManager, gammas.slotManager}};

      // --------------------------
      // *** Particle injection ***
      // --------------------------
      if (gpuState.injectState == InjectState::Idle) {
        for (auto &eventState : fEventStates) {
          if (const auto state = eventState.load(std::memory_order_acquire); state == EventState::G4RequestsFlush) {
            eventState = EventState::Inject;
          } else if (state == EventState::Inject) {
            eventState = EventState::InjectionCompleted;
          }
        }

        if (auto &toDevice = fBuffer->getActiveBuffer(); toDevice.nTrack > 0) {
          gpuState.injectState = InjectState::CreatingSlots;

          fBuffer->swapToDeviceBuffers();
          std::scoped_lock lock{toDevice.mutex};
          const auto nInject = std::min(toDevice.nTrack.load(), toDevice.maxTracks);
          toDevice.nTrack    = 0;

          if (fDebugLevel > 3) std::cout << "Injecting " << nInject << " to GPU\n";

          // copy buffer of tracks to device
          COPCORE_CUDA_CHECK(hipMemcpyAsync(gpuState.toDevice_dev.get(), toDevice.tracks,
                                             nInject * sizeof(adeptint::TrackData), hipMemcpyHostToDevice,
                                             transferStream));
          // Mark end of copy operation:
          COPCORE_CUDA_CHECK(hipEventRecord(cudaEvent, transferStream));

          // Inject AdePT tracks using the track buffer
          constexpr auto injectThreads = 128u;
          const auto injectBlocks      = (nInject + injectThreads - 1) / injectThreads;
          InjectTracks<<<injectBlocks, injectThreads, 0, transferStream>>>(gpuState.toDevice_dev.get(), nInject,
                                                                           secondaries, world_dev, fScoring_dev,
                                                                           gpuState.injectionQueue.get());
          COPCORE_CUDA_CHECK(hipLaunchHostFunc(
              transferStream,
              [](void *arg) { (*static_cast<decltype(GPUstate::injectState) *>(arg)) = InjectState::ReadyToEnqueue; },
              &gpuState.injectState));

          // Ensure that copy operation completed before releasing lock on to-device buffer
          COPCORE_CUDA_CHECK(hipEventSynchronize(cudaEvent));
        } else {
          gpuState.injectState = InjectState::Idle;
        }
      }

      // *** Enqueue particles that are ready on the device ***
      if (gpuState.injectState == InjectState::ReadyToEnqueue) {
        gpuState.injectState = InjectState::Enqueueing;
        EnqueueTracks<<<1, 256, 0, gpuState.stream>>>(allParticleQueues, gpuState.injectionQueue.get());
        // New injection has to wait until particles are enqueued:
        waitForOtherStream(transferStream, gpuState.stream);
      } else if (gpuState.injectState == InjectState::Enqueueing) {
        gpuState.injectState = InjectState::Idle;
      }

      // ------------------
      // *** Transport ***
      // ------------------

      // *** ELECTRONS ***
      {
        const auto [threads, blocks] = computeThreadsAndBlocks(particlesInFlight[ParticleType::Electron]);
        TransportElectrons<AdeptScoring><<<blocks, threads, 0, electrons.stream>>>(
            electrons.tracks, electrons.queues.currentlyActive, secondaries, electrons.queues.nextActive,
            electrons.queues.leakedTracksCurrent, fScoring_dev);

        COPCORE_CUDA_CHECK(hipEventRecord(electrons.event, electrons.stream));
        COPCORE_CUDA_CHECK(hipStreamWaitEvent(gpuState.stream, electrons.event, 0));
      }

      // *** POSITRONS ***
      {
        const auto [threads, blocks] = computeThreadsAndBlocks(particlesInFlight[ParticleType::Positron]);
        TransportPositrons<AdeptScoring><<<blocks, threads, 0, positrons.stream>>>(
            positrons.tracks, positrons.queues.currentlyActive, secondaries, positrons.queues.nextActive,
            positrons.queues.leakedTracksCurrent, fScoring_dev);

        COPCORE_CUDA_CHECK(hipEventRecord(positrons.event, positrons.stream));
        COPCORE_CUDA_CHECK(hipStreamWaitEvent(gpuState.stream, positrons.event, 0));
      }

      // *** GAMMAS ***
      {
        const auto [threads, blocks] = computeThreadsAndBlocks(particlesInFlight[ParticleType::Gamma]);
        TransportGammas<AdeptScoring><<<blocks, threads, 0, gammas.stream>>>(
            gammas.tracks, gammas.queues.currentlyActive, secondaries, gammas.queues.nextActive,
            gammas.queues.leakedTracksCurrent, fScoring_dev, gpuState.gammaInteractions);

        constexpr unsigned int intThreads = 128;
        ApplyGammaInteractions<AdeptScoring><<<dim3(20, 3, 1), intThreads, 0, gammas.stream>>>(
            gammas.tracks, secondaries, gammas.queues.nextActive, fScoring_dev, gpuState.gammaInteractions);

        COPCORE_CUDA_CHECK(hipEventRecord(gammas.event, gammas.stream));
        COPCORE_CUDA_CHECK(hipStreamWaitEvent(gpuState.stream, gammas.event, 0));
      }

      // ---------------------------------------
      // *** Count detailed event statistics ***
      // ---------------------------------------
      const bool prepareDeviceStats = true || iteration % 5 == 4 || fDebugLevel >= 3;
      if (prepareDeviceStats) {
        AdvanceEventStates(EventState::Transporting, EventState::WaitingForTransportToFinish);
        AdvanceEventStates(EventState::InjectionCompleted, EventState::Transporting);

        // Reset all counters count the currently flying population
        ZeroEventCounters<<<1, 256, 0, statsStream>>>(gpuState.stats_dev);
        CountCurrentPopulation<<<ParticleType::NumParticleTypes, 128, 0, statsStream>>>(
            allParticleQueues, gpuState.stats_dev, tracksAndSlots);
        // Count leaked tracks. Note that new tracks might be added while/after we count:
        CountLeakedTracks<<<2 * ParticleType::NumParticleTypes, 128, 0, statsStream>>>(
            allParticleQueues, gpuState.stats_dev, tracksAndSlots);

        waitForOtherStream(gpuState.stream, statsStream);

        // Get results to host:
        COPCORE_CUDA_CHECK(
            hipMemcpyAsync(gpuState.stats, gpuState.stats_dev, sizeof(Stats), hipMemcpyDeviceToHost, statsStream));
        COPCORE_CUDA_CHECK(hipEventRecord(cudaStatsEvent, statsStream));
      }

      // -------------------------
      // *** Collect particles ***
      // -------------------------

      if (gpuState.extractState == ExtractState::Idle &&
          std::any_of(fEventStates.begin(), fEventStates.end(), [](const auto &eventState) {
            const auto state = eventState.load(std::memory_order_acquire);
            return EventState::NeedDeviceFlush <= state && state < EventState::DeviceFlushed;
          })) {
        gpuState.extractState = ExtractState::FreeingSlots;

        // There is two device buffers, which might both need to be flushed
        AdvanceEventStates(EventState::FirstFlush, EventState::SecondFlush);
        AdvanceEventStates(EventState::NeedDeviceFlush, EventState::FirstFlush);

        const AllLeaked allLeaked = {.leakedElectrons = {electrons.tracks, electrons.queues.leakedTracksCurrent,
                                                         electrons.queues.leakedTracksNext, electrons.slotManager},
                                     .leakedPositrons = {positrons.tracks, positrons.queues.leakedTracksCurrent,
                                                         positrons.queues.leakedTracksNext, positrons.slotManager},
                                     .leakedGammas    = {gammas.tracks, gammas.queues.leakedTracksCurrent,
                                                         gammas.queues.leakedTracksNext, gammas.slotManager}};

        // Ensure that transport that's writing to the old queues finishes before collecting leaked tracks
        for (auto const &event : {electrons.event, positrons.event, gammas.event}) {
          COPCORE_CUDA_CHECK(hipStreamWaitEvent(transferStream, event));
        }

        // Populate the staging buffer and copy to host
        constexpr unsigned int block_size = 128;
        const unsigned int grid_size      = (gpuState.fNumFromDevice + block_size - 1) / block_size;
        FillFromDeviceBuffer<<<grid_size, block_size, 0, transferStream>>>(allLeaked, gpuState.fromDevice_dev.get(),
                                                                           gpuState.fNumFromDevice);
        COPCORE_CUDA_CHECK(hipMemcpyFromSymbolAsync(gpuState.nFromDevice.get(), nFromDevice_dev, sizeof(unsigned int),
                                                     0, hipMemcpyDeviceToHost, transferStream));
        COPCORE_CUDA_CHECK(hipLaunchHostFunc(
            transferStream,
            [](void *arg) { (*static_cast<decltype(GPUstate::extractState) *>(arg)) = ExtractState::ReadyToCopy; },
            &gpuState.extractState));
        ClearQueues<<<1, 1, 0, transferStream>>>(electrons.queues.leakedTracksCurrent,
                                                 positrons.queues.leakedTracksCurrent,
                                                 gammas.queues.leakedTracksCurrent);

        electrons.queues.SwapLeakedQueue();
        positrons.queues.SwapLeakedQueue();
        gammas.queues.SwapLeakedQueue();
      }

      if (gpuState.extractState == ExtractState::ReadyToCopy) {
        gpuState.extractState = ExtractState::CopyToHost;
        COPCORE_CUDA_CHECK(hipMemcpyAsync(gpuState.fromDevice_host.get(), gpuState.fromDevice_dev.get(),
                                           (*gpuState.nFromDevice) * sizeof(TrackData), hipMemcpyDeviceToHost,
                                           transferStream));
        COPCORE_CUDA_CHECK(hipLaunchHostFunc(
            transferStream, [](void *thisPtr) { static_cast<AdeptIntegration *>(thisPtr)->ReturnTracksToG4(); }, this));
      }

      // -------------------------
      // *** Finish iteration ***
      // -------------------------

      // This kernel needs to wait that all of the above work (except for asynchronous particle transfer) is done.
      // Don't forget to synchronise any of the transport or event counting with it.
      FinishIteration<<<4, 32, 0, gpuState.stream>>>(allParticleQueues, gpuState.stats_dev, tracksAndSlots,
                                                     gpuState.gammaInteractions);

      // Free slots if one of the queues is half full
      if (gpuState.injectState != InjectState::CreatingSlots &&
          (std::any_of(std::cbegin(gpuState.stats->usedSlots), std::cend(gpuState.stats->usedSlots),
                       [this](unsigned int capacity) { return capacity >= fTrackCapacity / 2; }) ||
           iteration % 100 == 0)) {
        // Freeing of slots has to run exclusively
        waitForOtherStream(gpuState.stream, transferStream);
        FreeSlots<<<ParticleType::NumParticleTypes, 256, 0, gpuState.stream>>>(tracksAndSlots);
        waitForOtherStream(transferStream, gpuState.stream);
      }

      // *** Synchronise all but transfer stream with the end of this iteration ***
      COPCORE_CUDA_CHECK(hipEventRecord(cudaEvent, gpuState.stream));
      for (auto stream : {electrons.stream, positrons.stream, gammas.stream, statsStream}) {
        COPCORE_CUDA_CHECK(hipStreamWaitEvent(stream, cudaEvent));
      }

      // ------------------------------------------
      // *** Take decisions for next iterations ***
      // ------------------------------------------

      // *** Count particles in flight ***
      if (prepareDeviceStats) {
        inFlight  = 0;
        numLeaked = 0;
        COPCORE_CUDA_CHECK(hipEventSynchronize(cudaStatsEvent));
        for (int i = 0; i < ParticleType::NumParticleTypes; i++) {
          inFlight += gpuState.stats->inFlight[i];
          numLeaked += gpuState.stats->leakedTracks[i];
          particlesInFlight[i] = gpuState.stats->inFlight[i];
        }

        for (unsigned short threadId = 0; threadId < fNThread; ++threadId) {
          const auto state = fEventStates[threadId].load(std::memory_order_acquire);
          if (state == EventState::WaitingForTransportToFinish && gpuState.stats->perEventInFlight[threadId] == 0) {
            fEventStates[threadId] = EventState::NeedDeviceFlush;
          }
          if (EventState::NeedDeviceFlush <= state && state < EventState::LeakedTracksRetrieved &&
              gpuState.stats->perEventInFlight[threadId] != 0) {
            std::cerr << "ERROR thread " << threadId << " is in state " << static_cast<unsigned int>(state)
                      << " and occupancy is " << gpuState.stats->perEventInFlight[threadId] << "\n";
          }
        }
      }

      // *** Notify G4 workers if their events completed ***
      if (std::any_of(fEventStates.begin(), fEventStates.end(),
                      [](const EventState &state) { return state == EventState::DeviceFlushed; })) {
        fBuffer->cv_fromDevice.notify_all();
      }

      if (fDebugLevel >= 3) {
        std::cerr << inFlight << " in flight ";
        std::cerr << "(" << gpuState.stats->inFlight[ParticleType::Electron] << " "
                  << gpuState.stats->inFlight[ParticleType::Positron] << " "
                  << gpuState.stats->inFlight[ParticleType::Gamma] << "),\tSlots:("
                  << gpuState.stats->usedSlots[ParticleType::Electron] << " "
                  << gpuState.stats->usedSlots[ParticleType::Positron] << " "
                  << gpuState.stats->usedSlots[ParticleType::Gamma] << ")";
        std::cerr << ", " << numLeaked << " leaked."
                  << "\tInjectState: " << static_cast<unsigned int>(gpuState.injectState.load())
                  << "\tExtractState: " << static_cast<unsigned int>(gpuState.extractState.load());
        if (fDebugLevel >= 4) {
          std::cerr << "\n\tper event: ";
          for (unsigned int i = 0; i < fNThread; ++i) {
            std::cerr << i << ": " << gpuState.stats->perEventInFlight[i]
                      << " (s=" << static_cast<unsigned short>(fEventStates[i].load(std::memory_order_acquire))
                      << ")\t";
          }
        }
        std::cerr << std::endl;
      }

      // TODO: Write this per thread
      // Check if only charged particles are left that are looping.
      if (gpuState.stats->inFlight[ParticleType::Electron] == previousElectrons &&
          gpuState.stats->inFlight[ParticleType::Positron] == previousPositrons &&
          gpuState.stats->inFlight[ParticleType::Gamma] == 0) {
        loopingNo++;
      } else {
        previousElectrons = gpuState.stats->inFlight[ParticleType::Electron];
        previousPositrons = gpuState.stats->inFlight[ParticleType::Positron];
        loopingNo         = 0;
      }

#ifndef NDEBUG
      // *** Check slots ***
      if (gpuState.injectState != InjectState::CreatingSlots && gpuState.extractState != ExtractState::FreeingSlots) {
        AssertConsistencyOfSlotManagers<<<120, 256, 0, gpuState.stream>>>(slotMgrArray, ParticleType::NumParticleTypes);
        COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));
      }

#if false
      for (int i = 0; i < ParticleType::NumParticleTypes; ++i) {
        ParticleType &part = gpuState.particles[i];
        COPCORE_CUDA_CHECK(hipMemcpyAsync(&part.slotManager_host, part.slotManager, sizeof(SlotManager),
                                           hipMemcpyDefault, gpuState.stream));
      }
      COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));
      {
        unsigned int slotsUsed[3];
        unsigned int slotsMax[3];
        unsigned int slotsToFree[3];
        for (int i = 0; i < ParticleType::NumParticleTypes; ++i) {
          ParticleType &part = gpuState.particles[i];
          slotsUsed[i]       = part.slotManager_host.fSlotCounter - part.slotManager_host.fFreeCounter;
          slotsMax[i]        = part.slotManager_host.fSlotCounterMax;
          slotsToFree[i]     = part.slotManager_host.fFreeCounter;
        }
        std::cout << "SlotManager: (" << slotsUsed[0] << ", " << slotsUsed[1] << ", " << slotsUsed[2]
                  << ") slots used.\ttoFree: (" << slotsToFree[0] << ", " << slotsToFree[1] << ", " << slotsToFree[2]
                  << ")\tmax: (" << slotsMax[0] << ", " << slotsMax[1] << ", " << slotsMax[2] << ")\n ";
      }
#endif
#endif
    }

    // TODO: Add special treatment of looping tracks

    AllParticleQueues queues = {{electrons.queues, positrons.queues, gammas.queues}};
    ClearAllQueues<<<1, 1, 0, gpuState.stream>>>(queues);
    COPCORE_CUDA_CHECK(hipStreamSynchronize(gpuState.stream));

    // TODO: This should be per event
    fScoring[0].fGlobalScoring.numKilled += inFlight;

    if (fDebugLevel > 2) std::cout << "End transport loop.\n";
  }
}

// SPDX-FileCopyrightText: 2023 CERN
// SPDX-License-Identifier: Apache-2.0
#include "ResourceManagement.h"

#include "CopCore/Global.h"

namespace adeptint {

void cudaDeleter(void *ptr)
{
  COPCORE_CUDA_CHECK(hipFree(ptr));
}

void cudaHostDeleter(void *ptr)
{
  COPCORE_CUDA_CHECK(hipHostFree(ptr));
}

void cudaStreamDeleter(hipStream_t *stream)
{
  COPCORE_CUDA_CHECK(hipStreamDestroy(*stream));
}

void cudaEventDeleter(hipEvent_t *event)
{
  COPCORE_CUDA_CHECK(hipEventDestroy(*event));
}
using unique_ptr_cudaEvent = std::unique_ptr<hipEvent_t, void (*)(hipEvent_t *)>;

} // namespace adeptint
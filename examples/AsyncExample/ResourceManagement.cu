// SPDX-FileCopyrightText: 2023 CERN
// SPDX-License-Identifier: Apache-2.0
#include "ResourceManagement.h"

#include "AdePT/copcore/Global.h"

namespace AsyncAdePT {

void freeCuda(void *ptr)
{
  if (ptr) COPCORE_CUDA_CHECK(hipFree(ptr));
}

void freeCudaHost(void *ptr)
{
  if (ptr) COPCORE_CUDA_CHECK(hipHostFree(ptr));
}

void freeCudaStream(void *stream)
{
  if (stream) COPCORE_CUDA_CHECK(hipStreamDestroy(*static_cast<hipStream_t *>(stream)));
}

void freeCudaEvent(void *event)
{
  if (event) COPCORE_CUDA_CHECK(hipEventDestroy(*static_cast<hipEvent_t *>(event)));
}

} // namespace AsyncAdePT

// SPDX-FileCopyrightText: 2023 CERN
// SPDX-License-Identifier: Apache-2.0
#include "ResourceManagement.h"

#include "AdePT/copcore/Global.h"

namespace AsyncAdePT {

void cudaDeleter(void *ptr)
{
  COPCORE_CUDA_CHECK(hipFree(ptr));
}

void cudaHostDeleter(void *ptr)
{
  COPCORE_CUDA_CHECK(hipHostFree(ptr));
}

void cudaStreamDeleter(void *stream)
{
  COPCORE_CUDA_CHECK(hipStreamDestroy(*static_cast<hipStream_t *>(stream)));
}

void cudaEventDeleter(void *event)
{
  COPCORE_CUDA_CHECK(hipEventDestroy(*static_cast<hipEvent_t *>(event)));
}

} // namespace AsyncAdePT

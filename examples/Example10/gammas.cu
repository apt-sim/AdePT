#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2021 CERN
// SPDX-License-Identifier: Apache-2.0

#include "example10.cuh"

#include <AdePT/LoopNavigator.h>
#include <CopCore/PhysicalConstants.h>

#include <G4HepEmGammaManager.hh>
#include <G4HepEmGammaTrack.hh>
#include <G4HepEmTrack.hh>
#include <G4HepEmGammaInteractionCompton.hh>
#include <G4HepEmGammaInteractionConversion.hh>
#include <G4HepEmGammaInteractionPhotoelectric.hh>
// Pull in implementation.
#include <G4HepEmGammaManager.icc>
#include <G4HepEmGammaInteractionCompton.icc>
#include <G4HepEmGammaInteractionConversion.icc>
#include <G4HepEmGammaInteractionPhotoelectric.icc>

__global__ void TransportGammas(Track *gammas, const adept::MParray *active, Secondaries secondaries,
                                adept::MParray *activeQueue, adept::MParray *relocateQueue, GlobalScoring *scoring,
                                int maxSteps)
{
  int activeSize = active->size();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < activeSize; i += blockDim.x * gridDim.x) {
    const int slot      = (*active)[i];
    Track &currentTrack = gammas[slot];

    // Init a track with the needed data to call into G4HepEm.
    G4HepEmGammaTrack gammaTrack;
    G4HepEmTrack *theTrack = gammaTrack.GetTrack();
    theTrack->SetEKin(currentTrack.energy);
    // For now, just assume a single material.
    int theMCIndex = 1;
    theTrack->SetMCIndex(theMCIndex);

    bool alive = true;
    for (int s = 0; alive && s < maxSteps; s++) {

      // Sample the `number-of-interaction-left` and put it into the track.
      for (int ip = 0; ip < 3; ++ip) {
        double numIALeft = currentTrack.numIALeft[ip];
        if (numIALeft <= 0) {
          numIALeft                  = -std::log(currentTrack.Uniform());
          currentTrack.numIALeft[ip] = numIALeft;
        }
        theTrack->SetNumIALeft(numIALeft, ip);
      }

      // Call G4HepEm to compute the physics step limit.
      G4HepEmGammaManager::HowFar(&g4HepEmData, &g4HepEmPars, &gammaTrack);

      // Get result into variables.
      double geometricalStepLengthFromPhysics = theTrack->GetGStepLength();
      int winnerProcessIndex                  = theTrack->GetWinnerProcessIndex();
      // Leave the range and MFP inside the G4HepEmTrack. If we split kernels, we
      // also need to carry them over!

      // Check if there's a volume boundary in between.
      double geometryStepLength =
          LoopNavigator::ComputeStepAndNextVolume(currentTrack.pos, currentTrack.dir, geometricalStepLengthFromPhysics,
                                                  currentTrack.currentState, currentTrack.nextState);
      currentTrack.pos += geometryStepLength * currentTrack.dir;

      if (currentTrack.nextState.IsOnBoundary()) {
        theTrack->SetGStepLength(geometryStepLength);
        theTrack->SetOnBoundary(true);
      }

      G4HepEmGammaManager::UpdateNumIALeft(theTrack);

      // Save the `number-of-interaction-left` in our track.
      for (int ip = 0; ip < 3; ++ip) {
        double numIALeft           = theTrack->GetNumIALeft(ip);
        currentTrack.numIALeft[ip] = numIALeft;
      }

      if (currentTrack.nextState.IsOnBoundary()) {
        // For now, just count that we hit something.
        atomicAdd(&scoring->hits, 1);

        // Kill the particle if it left the world.
        if (currentTrack.nextState.Top() != nullptr) {
          alive = true;
          relocateQueue->push_back(slot);

          // Move to the next boundary.
          currentTrack.SwapStates();
        } else {
          alive = false;
        }

        // Cannot continue for now: either the particles left the world, or we
        // need to relocate it to the next volume.
        break;
      } else if (winnerProcessIndex < 0) {
        // No discrete process, move on.
        continue;
      }

      // Reset number of interaction left for the winner discrete process.
      // (Will be resampled in the next iteration.)
      currentTrack.numIALeft[winnerProcessIndex] = -1.0;

      // Perform the discrete interaction.
      RanluxppDoubleEngine rnge(&currentTrack.rngState);
      // We might need one branched RNG state, prepare while threads are synchronized.
      RanluxppDouble newRNG(currentTrack.rngState.Branch());

      const double energy = currentTrack.energy;

      switch (winnerProcessIndex) {
      case 0: {
        // Invoke gamma conversion to e-/e+ pairs, if the energy is above the threshold.
        if (energy < 2 * copcore::units::kElectronMassC2) {
          alive = true;
          continue;
        }

        double logEnergy = std::log(energy);
        double elKinEnergy, posKinEnergy;
        G4HepEmGammaInteractionConversion::SampleKinEnergies(&g4HepEmData, energy, logEnergy, theMCIndex, elKinEnergy,
                                                             posKinEnergy, &rnge);

        double dirPrimary[] = {currentTrack.dir.x(), currentTrack.dir.y(), currentTrack.dir.z()};
        double dirSecondaryEl[3], dirSecondaryPos[3];
        G4HepEmGammaInteractionConversion::SampleDirections(dirPrimary, dirSecondaryEl, dirSecondaryPos, elKinEnergy,
                                                            posKinEnergy, &rnge);

        Track &electron = secondaries.electrons.NextTrack();
        Track &positron = secondaries.positrons.NextTrack();
        atomicAdd(&scoring->secondaries, 2);

        electron.InitAsSecondary(/*parent=*/currentTrack);
        electron.rngState = newRNG;
        electron.energy   = elKinEnergy;
        electron.dir.Set(dirSecondaryEl[0], dirSecondaryEl[1], dirSecondaryEl[2]);

        positron.InitAsSecondary(/*parent=*/currentTrack);
        // Reuse the RNG state of the dying track.
        positron.rngState = currentTrack.rngState;
        positron.energy   = posKinEnergy;
        positron.dir.Set(dirSecondaryPos[0], dirSecondaryPos[1], dirSecondaryPos[2]);

        alive = false;
        break;
      }
      case 1: {
        // Invoke Compton scattering of gamma.
        constexpr double LowEnergyThreshold = 100 * copcore::units::eV;
        if (energy < LowEnergyThreshold) {
          alive = true;
          continue;
        }
        const double origDirPrimary[] = {currentTrack.dir.x(), currentTrack.dir.y(), currentTrack.dir.z()};
        double dirPrimary[3];
        const double newEnergyGamma =
            G4HepEmGammaInteractionCompton::SamplePhotonEnergyAndDirection(energy, dirPrimary, origDirPrimary, &rnge);
        vecgeom::Vector3D<double> newDirGamma(dirPrimary[0], dirPrimary[1], dirPrimary[2]);

        const double energyEl = energy - newEnergyGamma;
        if (energyEl > LowEnergyThreshold) {
          // Create a secondary electron and sample/compute directions.
          Track &electron = secondaries.electrons.NextTrack();
          atomicAdd(&scoring->secondaries, 1);

          electron.InitAsSecondary(/*parent=*/currentTrack);
          electron.rngState = newRNG;
          electron.energy   = energyEl;
          electron.dir      = energy * currentTrack.dir - newEnergyGamma * newDirGamma;
          electron.dir.Normalize();
        } else {
          atomicAdd(&scoring->energyDeposit, energyEl);
        }

        // Check the new gamma energy and deposit if below threshold.
        if (newEnergyGamma > LowEnergyThreshold) {
          currentTrack.energy = newEnergyGamma;
          theTrack->SetEKin(currentTrack.energy);
          currentTrack.dir = newDirGamma;

          // The current track continues to live.
          alive = true;
        } else {
          alive = false;
          atomicAdd(&scoring->energyDeposit, newEnergyGamma);
        }
        break;
      }
      case 2: {
        // Invoke photoelectric process.
        const double theLowEnergyThreshold = 1 * copcore::units::eV;

        const double bindingEnergy = G4HepEmGammaInteractionPhotoelectric::SelectElementBindingEnergy(
            &g4HepEmData, theMCIndex, gammaTrack.GetPEmxSec(), energy, &rnge);

        double edep             = bindingEnergy;
        const double photoElecE = energy - edep;
        if (photoElecE > theLowEnergyThreshold) {
          // Create a secondary electron and sample directions.
          Track &electron = secondaries.electrons.NextTrack();
          atomicAdd(&scoring->secondaries, 1);

          double dirGamma[] = {currentTrack.dir.x(), currentTrack.dir.y(), currentTrack.dir.z()};
          double dirPhotoElec[3];
          G4HepEmGammaInteractionPhotoelectric::SamplePhotoElectronDirection(photoElecE, dirGamma, dirPhotoElec, &rnge);

          electron.InitAsSecondary(/*parent=*/currentTrack);
          electron.rngState = newRNG;
          electron.energy   = photoElecE;
          electron.dir.Set(dirPhotoElec[0], dirPhotoElec[1], dirPhotoElec[2]);
        } else {
          edep = energy;
        }
        atomicAdd(&scoring->energyDeposit, edep);
        alive = false;
        break;
      }
      }
    }

    if (alive) {
      activeQueue->push_back(slot);
    }
  }
}

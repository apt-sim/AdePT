#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2020 CERN
// SPDX-License-Identifier: Apache-2.0

#include <CopCore/Ranluxpp.h>

#include <iostream>

__global__ void kernel(RanluxppDouble *r, double *d, uint64_t *i, double *d2)
{
  *d = r->Rndm();
  *i = r->IntRndm();
  r->Skip(42);
  *d2 = r->Rndm();
}

int main(void)
{
  RanluxppDouble r;
  std::cout << "double: " << r.Rndm() << std::endl;
  std::cout << "int: " << r.IntRndm() << std::endl;

  RanluxppDouble *r_dev;
  hipMalloc(&r_dev, sizeof(RanluxppDouble));
  double *d_dev_ptr;
  uint64_t *i_dev_ptr;
  double *d2_dev_ptr;
  hipMalloc(&d_dev_ptr, sizeof(double));
  hipMalloc(&i_dev_ptr, sizeof(uint64_t));
  hipMalloc(&d2_dev_ptr, sizeof(double));

  // Transfer the state of the generator to the device.
  hipMemcpy(r_dev, &r, sizeof(RanluxppDouble), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  kernel<<<1, 1>>>(r_dev, d_dev_ptr, i_dev_ptr, d2_dev_ptr);
  hipDeviceSynchronize();

  // Generate from the same state on the host.
  double d   = r.Rndm();
  uint64_t i = r.IntRndm();
  r.Skip(42);
  double d2 = r.Rndm();

  // Fetch the numbers from the device for comparison.
  double d_dev;
  uint64_t i_dev;
  double d2_dev;
  hipMemcpy(&d_dev, d_dev_ptr, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&i_dev, i_dev_ptr, sizeof(uint64_t), hipMemcpyDeviceToHost);
  hipMemcpy(&d2_dev, d2_dev_ptr, sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  int ret = 0;

  std::cout << std::endl;
  std::cout << "double:" << std::endl;
  std::cout << "   host:   " << d << std::endl;
  std::cout << "   device: " << d_dev << std::endl;
  ret += (d != d_dev);

  std::cout << "int:" << std::endl;
  std::cout << "   host:   " << i << std::endl;
  std::cout << "   device: " << i_dev << std::endl;
  ret += (i != i_dev);

  std::cout << "double (after calling Skip(42)):" << std::endl;
  std::cout << "   host:   " << d2 << std::endl;
  std::cout << "   device: " << d2_dev << std::endl;
  ret += (d2 != d2_dev);

  hipFree(r_dev);
  hipFree(d_dev_ptr);
  hipFree(i_dev_ptr);
  hipFree(d2_dev_ptr);

  return ret;
}
